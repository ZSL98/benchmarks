#include "shared.h"
// Node name:	Constant_505
// Description:	Constant
// Input:
// Output:
//	- name: Constant_505_0	type: float	shape: Shape{2048, 1024, 1, 1}
void Constant_float_cuda_Constant_505(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_505_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_505_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8388608];
    bin_file.read(tmp_mem, 8388608);
    hipMemcpyAsync(output0, tmp_mem, 8388608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

