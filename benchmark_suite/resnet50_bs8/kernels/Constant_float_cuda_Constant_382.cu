#include "shared.h"
// Node name:	Constant_382
// Description:	Constant
// Input:
// Output:
//	- name: Constant_382_0	type: float	shape: Shape{64, 64, 3, 3}
void Constant_float_cuda_Constant_382(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_382_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_382_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[147456];
    bin_file.read(tmp_mem, 147456);
    hipMemcpyAsync(output0, tmp_mem, 147456, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

