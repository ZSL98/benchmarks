#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_547
// Description:	Matched_Pattern
// Input:
//	- name: Relu_179_0	type: float	shape: Shape{8, 512, 28, 28}
//	- name: Constant_445_0	type: float	shape: Shape{256, 512, 1, 1}
//	- name: Broadcast_446_0	type: float	shape: Shape{8, 256, 28, 28}
// Output:
//	- name: Relu_188_0	type: float	shape: Shape{8, 256, 28, 28}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_547(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[896];
    __shared__ float input1_shared[1024];
    {
        float* compute = output0;
        {
          float conv2d_nchw[16];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[14] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          conv2d_nchw[15] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
            __syncthreads();
            *(float2*)(pad_temp_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input0 + ((((((((int)blockIdx.x) / 56) * 401408) + (rc_outer_outer * 12544)) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 28) * 2)));
            *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 448)) = *(float2*)(input0 + (((((((((int)blockIdx.x) / 56) * 401408) + (rc_outer_outer * 12544)) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 28) * 2)) + 6272));
            input1_shared[(((int)threadIdx.x) * 2)] = input1[((((((((int)blockIdx.x) % 56) / 14) * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2))];
            input1_shared[((((int)threadIdx.x) * 2) + 1)] = input1[(((((((((int)blockIdx.x) % 56) / 14) * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 1)];
            input1_shared[((((int)threadIdx.x) * 2) + 448)] = input1[(((((((((int)blockIdx.x) % 56) / 14) * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 14336)];
            input1_shared[((((int)threadIdx.x) * 2) + 449)] = input1[(((((((((int)blockIdx.x) % 56) / 14) * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 14337)];
            if (((int)threadIdx.x) < 64) {
              input1_shared[((((int)threadIdx.x) * 2) + 896)] = input1[(((((((((int)blockIdx.x) % 56) / 14) * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 28672)];
              input1_shared[((((int)threadIdx.x) * 2) + 897)] = input1[(((((((((int)blockIdx.x) % 56) / 14) * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 28673)];
            }
            __syncthreads();
            for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
              for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
                conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[(((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2))]));
                conv2d_nchw[(xx_outer_inner + 2)] = (conv2d_nchw[(xx_outer_inner + 2)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 28)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2))]));
                conv2d_nchw[(xx_outer_inner + 4)] = (conv2d_nchw[(xx_outer_inner + 4)] + (pad_temp_shared[(((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 256)]));
                conv2d_nchw[(xx_outer_inner + 6)] = (conv2d_nchw[(xx_outer_inner + 6)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 28)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 256)]));
                conv2d_nchw[(xx_outer_inner + 8)] = (conv2d_nchw[(xx_outer_inner + 8)] + (pad_temp_shared[(((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 512)]));
                conv2d_nchw[(xx_outer_inner + 10)] = (conv2d_nchw[(xx_outer_inner + 10)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 28)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 512)]));
                conv2d_nchw[(xx_outer_inner + 12)] = (conv2d_nchw[(xx_outer_inner + 12)] + (pad_temp_shared[(((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 768)]));
                conv2d_nchw[(xx_outer_inner + 14)] = (conv2d_nchw[(xx_outer_inner + 14)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 28)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 768)]));
                conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 56)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 1)]));
                conv2d_nchw[(xx_outer_inner + 2)] = (conv2d_nchw[(xx_outer_inner + 2)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 84)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 1)]));
                conv2d_nchw[(xx_outer_inner + 4)] = (conv2d_nchw[(xx_outer_inner + 4)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 56)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 257)]));
                conv2d_nchw[(xx_outer_inner + 6)] = (conv2d_nchw[(xx_outer_inner + 6)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 84)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 257)]));
                conv2d_nchw[(xx_outer_inner + 8)] = (conv2d_nchw[(xx_outer_inner + 8)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 56)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 513)]));
                conv2d_nchw[(xx_outer_inner + 10)] = (conv2d_nchw[(xx_outer_inner + 10)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 84)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 513)]));
                conv2d_nchw[(xx_outer_inner + 12)] = (conv2d_nchw[(xx_outer_inner + 12)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 56)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 769)]));
                conv2d_nchw[(xx_outer_inner + 14)] = (conv2d_nchw[(xx_outer_inner + 14)] + (pad_temp_shared[((((rc_outer_inner * 112) + ((((int)threadIdx.x) % 14) * 2)) + xx_outer_inner) + 84)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 2)) + 769)]));
              }
            }
          }
          for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
            compute[((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner)] = max((conv2d_nchw[i3_inner] + input2[((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 28)] = max((conv2d_nchw[(i3_inner + 2)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 28)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 12544)] = max((conv2d_nchw[(i3_inner + 4)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 12544)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 12572)] = max((conv2d_nchw[(i3_inner + 6)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 12572)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 25088)] = max((conv2d_nchw[(i3_inner + 8)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 25088)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 25116)] = max((conv2d_nchw[(i3_inner + 10)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 25116)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 37632)] = max((conv2d_nchw[(i3_inner + 12)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 37632)]), 0.000000e+00f);
            compute[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 37660)] = max((conv2d_nchw[(i3_inner + 14)] + input2[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner) + 37660)]), 0.000000e+00f);
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_547_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_547<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

