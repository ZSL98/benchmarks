#include "shared.h"
// Node name:	Constant_406
// Description:	Constant
// Input:
// Output:
//	- name: Constant_406_0	type: float	shape: Shape{128, 256, 1, 1}
void Constant_float_cuda_Constant_406(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_406_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_406_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

