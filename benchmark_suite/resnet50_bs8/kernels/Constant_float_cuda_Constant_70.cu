#include "shared.h"
// Node name:	Constant_70
// Description:	Constant
// Input:
// Output:
//	- name: Constant_70_0	type: float	shape: Shape{2048, 1000}
void Constant_float_cuda_Constant_70(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_70_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_70_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8192000];
    bin_file.read(tmp_mem, 8192000);
    hipMemcpyAsync(output0, tmp_mem, 8192000, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

