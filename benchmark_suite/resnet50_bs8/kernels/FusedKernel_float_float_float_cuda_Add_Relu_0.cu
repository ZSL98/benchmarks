#include "hip/hip_runtime.h"
#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: BatchNormInference_92_0	type: float	shape: Shape{8, 256, 56, 56}
//	- name: BatchNormInference_83_0	type: float	shape: Shape{8, 256, 56, 56}
// Output:
//	- name: Relu_94_0	type: float	shape: Shape{8, 256, 56, 56}
// Fused functions:
// Add, resnet_model/add
// Relu, resnet_model/Relu_3
extern "C" __launch_bounds__(128) __global__ void FusedKernel_float_float_float_cuda_Add_Relu_0(float* input0, float* input1, float* output0)
{
    int tid = blockIdx.x * 128 * 2 + threadIdx.x;
    int tid1 = blockIdx.x * 128* 2 + threadIdx.x + 128;
    float temp0 = add(input0[tid], input1[tid]);
    float temp0_1 = add(input0[tid1], input1[tid1]);
    float temp1 = relu(temp0);
    float temp1_1 = relu(temp0_1);
    output0[tid] = temp1;
    output0[tid1] = temp1_1;

}
extern void FusedKernel_float_float_float_cuda_Add_Relu_0_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    FusedKernel_float_float_float_cuda_Add_Relu_0<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

