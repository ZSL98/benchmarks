#include "shared.h"
// Node name:	Constant_397
// Description:	Constant
// Input:
// Output:
//	- name: Constant_397_0	type: float	shape: Shape{64, 256, 1, 1}
void Constant_float_cuda_Constant_397(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_397_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_397_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

