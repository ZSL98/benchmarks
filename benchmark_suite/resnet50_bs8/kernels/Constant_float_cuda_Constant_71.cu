#include "shared.h"
// Node name:	Constant_71
// Description:	Constant
// Input:
// Output:
//	- name: Constant_71_0	type: float	shape: Shape{1000}
void Constant_float_cuda_Constant_71(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_71_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_71_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4000];
    bin_file.read(tmp_mem, 4000);
    hipMemcpyAsync(output0, tmp_mem, 4000, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

