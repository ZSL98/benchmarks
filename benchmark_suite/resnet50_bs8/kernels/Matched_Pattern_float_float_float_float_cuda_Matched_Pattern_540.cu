#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_540
// Description:	Matched_Pattern
// Input:
//	- name: Pad_131_0	type: float	shape: Shape{8, 128, 58, 58}
//	- name: Constant_412_0	type: float	shape: Shape{128, 128, 3, 3}
//	- name: Broadcast_413_0	type: float	shape: Shape{8, 128, 28, 28}
// Output:
//	- name: Relu_135_0	type: float	shape: Shape{8, 128, 28, 28}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_540(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[162];
    __shared__ float input1_shared[2304];
    {
        float* compute = output0;
        {
          float conv2d_nchw[16];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[14] = 0.000000e+00f;
          conv2d_nchw[15] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
            __syncthreads();
            pad_temp_shared[((int)threadIdx.x)] = input0[((((((((((int)blockIdx.x) / 49) * 430592) + (rc_outer_outer * 6728)) + ((((int)threadIdx.x) / 81) * 3364)) + (((((int)blockIdx.x) % 49) / 7) * 464)) + (((((int)threadIdx.x) % 81) / 9) * 58)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) % 9))];
            if (((int)threadIdx.x) < 34) {
              pad_temp_shared[(((int)threadIdx.x) + 128)] = input0[((((((((((int)blockIdx.x) / 49) * 430592) + (rc_outer_outer * 6728)) + (((((int)threadIdx.x) + 128) / 81) * 3364)) + (((((int)blockIdx.x) % 49) / 7) * 464)) + (((((int)threadIdx.x) + 47) / 9) * 58)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 2) % 9))];
            }
            *(float2*)(input1_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input1 + ((((((int)threadIdx.x) / 9) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) % 9) * 2)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 256) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 4) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 512) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 8) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 768) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 12) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 1024) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 16) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 1280) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 2) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 1536) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 6) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1792)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 1792) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 10) % 18)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 2048)) = *(float2*)(input1 + ((((((((int)threadIdx.x) * 2) + 2048) / 18) * 1152) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 14) % 18)));
            __syncthreads();
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 4)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 2)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 4)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 2)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 1)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 3)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 1)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 3)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 2)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 4)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 2)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 4)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 81)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 83)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 81)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 83)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 82)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 84)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 82)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 84)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 83)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 85)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 83)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 85)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 18)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 18)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 19)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 21)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 19)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 21)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 22)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 22)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 99)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 99)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 100)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 102)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 100)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 102)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 103)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 103)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 36)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 36)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 37)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 39)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 37)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 39)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 40)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 40)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 117)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 117)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 118)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 120)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 118)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 120)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 121)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 121)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 54)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[((((int)threadIdx.x) >> 1) * 36)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 54)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 55)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 57)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 55)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 57)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 58)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 58)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 135)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 135)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 136)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 138)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 136)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 138)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 139)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 139)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 9)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 11)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 9)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 11)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 10)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 12)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 10)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 12)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 11)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 13)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 11)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 13)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 90)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 92)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 90)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 92)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 91)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 93)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 91)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 93)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 92)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 94)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 92)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 94)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 27)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 29)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 27)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 29)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 28)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 30)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 28)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 30)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 29)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 31)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 29)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 31)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 108)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 110)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 108)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 110)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 109)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 111)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 109)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 111)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 110)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 112)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 110)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 112)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 45)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 47)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 45)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 47)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 46)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 48)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 46)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 48)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 47)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 49)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 47)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 49)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 126)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 128)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 126)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 128)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 127)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 129)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 127)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 129)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 128)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 130)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 128)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 130)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 63)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 65)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 63)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 65)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 64)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 66)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 64)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 66)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 65)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 67)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 65)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 67)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 144)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 146)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 144)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 146)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 145)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 147)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 145)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 147)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 146)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 148)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 146)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 148)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 18)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 18)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 19)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 21)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 19)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 21)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 22)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 20)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 22)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 99)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 99)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 100)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 102)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 100)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 102)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 103)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 101)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 103)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 36)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 36)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 37)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 39)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 37)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 39)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 40)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 38)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 40)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 117)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 117)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 118)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 120)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 118)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 120)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 121)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 119)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 121)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 54)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 54)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 55)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 57)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 55)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 57)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 58)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 56)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 58)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 135)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 135)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 136)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 138)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 136)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 138)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 139)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 137)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 139)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 72)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 74)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 72)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 74)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 73)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 75)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 73)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 75)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 74)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 76)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 74)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 76)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 153)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 155)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 153)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 155)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 154)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 156)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 154)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 156)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 155)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 157)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 155)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 4) + 157)] * input1_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
          }
          for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
            for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
              for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
                compute[(((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 1) * 1568)) + (i1_inner * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + i3_inner)] = max((conv2d_nchw[(((i1_inner * 8) + (i2_inner * 2)) + i3_inner)] + input2[(((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 1) * 1568)) + (i1_inner * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + i3_inner)]), 0.000000e+00f);
              }
            }
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_540_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_540<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

