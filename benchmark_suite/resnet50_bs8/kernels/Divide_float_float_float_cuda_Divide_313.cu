#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Divide_313
// Description:	Divide
// Input:
//	- name: Sum_311_0	type: float	shape: Shape{8, 2048}
//	- name: Constant_312_0	type: float	shape: Shape{8, 2048}
// Output:
//	- name: Divide_313_0	type: float	shape: Shape{8, 2048}
extern "C" __launch_bounds__(128) __global__ void Divide_float_float_float_cuda_Divide_313(float* input0, float* input1, float* output0)
{
    output0[blockIdx.x * 128 * 2 + threadIdx.x] = fdividef(input0[blockIdx.x * 128 * 2 + threadIdx.x], input1[blockIdx.x * 128 * 2 + threadIdx.x]);
    output0[blockIdx.x * 128 * 2 + threadIdx.x + 128] = fdividef(input0[blockIdx.x * 128 * 2 + threadIdx.x + 128], input1[blockIdx.x * 128 * 2 + threadIdx.x + 128]);

}
extern void Divide_float_float_float_cuda_Divide_313_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Divide_float_float_float_cuda_Divide_313<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

