#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_539
// Description:	Matched_Pattern
// Input:
//	- name: Relu_120_0	type: float	shape: Shape{8, 256, 56, 56}
//	- name: Constant_406_0	type: float	shape: Shape{128, 256, 1, 1}
//	- name: Broadcast_407_0	type: float	shape: Shape{8, 128, 56, 56}
// Output:
//	- name: Relu_129_0	type: float	shape: Shape{8, 128, 56, 56}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_539(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[1024];
    __shared__ float input1_shared[4096];
    {
        float* compute = output0;
        {
          float conv2d_nchw[16];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          conv2d_nchw[14] = 0.000000e+00f;
          conv2d_nchw[15] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
            __syncthreads();
            *(float2*)(pad_temp_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input0 + ((((((((((int)blockIdx.x) / 98) * 802816) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 4) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 2) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)));
            *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(input0 + (((((((((((int)blockIdx.x) / 98) * 802816) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 4) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 2) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 50176));
            *(float2*)(input1_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input1 + ((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 4096));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8192));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 12288));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 2048)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 16384));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 2560)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 20480));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 3072)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 24576));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 3584)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 28672));
            __syncthreads();
            for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
              conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
              conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
              conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
              conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
              conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
              conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
              conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
              conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
              conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
              conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
              conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
              conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
              conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
              conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
              conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2))] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
              conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
            }
          }
          for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
            for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
              compute[(((((((((((int)blockIdx.x) / 98) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 2) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner)] = max((conv2d_nchw[((i1_inner * 2) + i3_inner)] + input2[(((((((((((int)blockIdx.x) / 98) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 2) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner)]), 0.000000e+00f);
            }
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_539_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_539<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

