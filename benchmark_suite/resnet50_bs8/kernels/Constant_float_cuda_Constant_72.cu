#include "shared.h"
// Node name:	Constant_72
// Description:	Constant
// Input:
// Output:
//	- name: Constant_72_0	type: float	shape: Shape{}
void Constant_float_cuda_Constant_72(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_72_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_72_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4];
    bin_file.read(tmp_mem, 4);
    hipMemcpyAsync(output0, tmp_mem, 4, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

