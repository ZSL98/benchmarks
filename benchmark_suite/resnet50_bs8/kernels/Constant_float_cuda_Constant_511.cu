#include "shared.h"
// Node name:	Constant_511
// Description:	Constant
// Input:
// Output:
//	- name: Constant_511_0	type: float	shape: Shape{2048, 512, 1, 1}
void Constant_float_cuda_Constant_511(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_511_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_511_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4194304];
    bin_file.read(tmp_mem, 4194304);
    hipMemcpyAsync(output0, tmp_mem, 4194304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

