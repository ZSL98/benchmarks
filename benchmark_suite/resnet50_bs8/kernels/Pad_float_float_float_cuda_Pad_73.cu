#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Pad_73
// Description:	Pad
// Input:
//	- name: Parameter_30_0	type: float	shape: Shape{8, 3, 224, 224}
//	- name: Constant_72_0	type: float	shape: Shape{}
// Output:
//	- name: Pad_73_0	type: float	shape: Shape{8, 3, 230, 230}
extern "C" __launch_bounds__(64) __global__ void Pad_float_float_float_cuda_Pad_73(float* input0, float* input1, float* output0)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float* in = input0;
    float* pad = input1;
    float* out = output0;
    if (tid < 1269600)
    {
        size_t input_shape0 = 8;
        size_t input_shape1 = 3;
        size_t input_shape2 = 224;
        size_t input_shape3 = 224;
        uint32_t input_strides0 = 150528;
        uint32_t input_strides1 = 50176;
        uint32_t input_strides2 = 224;
        uint32_t input_strides3 = 1;
        uint32_t output_strides0 = 158700;
        uint32_t output_strides1 = 52900;
        uint32_t output_strides2 = 230;
        uint32_t output_strides3 = 1;
        uint32_t padding_below0 = 0;
        uint32_t padding_below1 = 0;
        uint32_t padding_below2 = 3;
        uint32_t padding_below3 = 3;
        uint32_t padding_interior0 = 0;
        uint32_t padding_interior1 = 0;
        uint32_t padding_interior2 = 0;
        uint32_t padding_interior3 = 0;
        bool in_bounds = true;
        uint32_t output_pixel = tid;
        uint32_t input_pixel = 0;
        int32_t input, input_dil;
        input_dil = output_pixel / output_strides0 - padding_below0;
        input = input_dil / (padding_interior0 + 1);
        input_dil %= (padding_interior0 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
        input_pixel += input * input_strides0;
        output_pixel %= output_strides0;
        input_dil = output_pixel / output_strides1 - padding_below1;
        input = input_dil / (padding_interior1 + 1);
        input_dil %= (padding_interior1 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
        input_pixel += input * input_strides1;
        output_pixel %= output_strides1;
        input_dil = output_pixel / output_strides2 - padding_below2;
        input = input_dil / (padding_interior2 + 1);
        input_dil %= (padding_interior2 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
        input_pixel += input * input_strides2;
        output_pixel %= output_strides2;
        input_dil = output_pixel / output_strides3 - padding_below3;
        input = input_dil / (padding_interior3 + 1);
        input_dil %= (padding_interior3 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
        input_pixel += input * input_strides3;
        out[tid] = (in_bounds) ? in[input_pixel] : *pad;
    }

}
extern void Pad_float_float_float_cuda_Pad_73_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Pad_float_float_float_cuda_Pad_73<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

