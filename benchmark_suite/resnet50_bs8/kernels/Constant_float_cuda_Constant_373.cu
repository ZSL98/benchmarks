#include "shared.h"
// Node name:	Constant_373
// Description:	Constant
// Input:
// Output:
//	- name: Constant_373_0	type: float	shape: Shape{64, 3, 7, 7}
void Constant_float_cuda_Constant_373(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_373_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_373_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[37632];
    bin_file.read(tmp_mem, 37632);
    hipMemcpyAsync(output0, tmp_mem, 37632, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

