#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_549
// Description:	Matched_Pattern
// Input:
//	- name: Relu_199_0	type: float	shape: Shape{8, 1024, 14, 14}
//	- name: Constant_457_0	type: float	shape: Shape{256, 1024, 1, 1}
//	- name: Broadcast_458_0	type: float	shape: Shape{8, 256, 14, 14}
// Output:
//	- name: Relu_203_0	type: float	shape: Shape{8, 256, 14, 14}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[256];
    __shared__ float input1_shared[1024];
    {
        float* compute = output0;
        {
          float conv2d_nchw[32];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[16] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[17] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[20] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[21] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[18] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[19] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[22] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[23] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[24] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[25] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[28] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          conv2d_nchw[29] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[26] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[27] = 0.000000e+00f;
          conv2d_nchw[14] = 0.000000e+00f;
          conv2d_nchw[30] = 0.000000e+00f;
          conv2d_nchw[15] = 0.000000e+00f;
          conv2d_nchw[31] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
            __syncthreads();
            *(float2*)(pad_temp_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input0 + (((((((((int)blockIdx.x) / 196) * 802816) + (rc_outer_outer * 3136)) + ((((int)threadIdx.x) >> 1) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)));
            *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 64)) = *(float2*)(input0 + ((((((((((int)blockIdx.x) / 196) * 802816) + (rc_outer_outer * 3136)) + ((((int)threadIdx.x) >> 1) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + 200704));
            *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 128)) = *(float2*)(input0 + ((((((((((int)blockIdx.x) / 196) * 802816) + (rc_outer_outer * 3136)) + ((((int)threadIdx.x) >> 1) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + 401408));
            *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 192)) = *(float2*)(input0 + ((((((((((int)blockIdx.x) / 196) * 802816) + (rc_outer_outer * 3136)) + ((((int)threadIdx.x) >> 1) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + 602112));
            *(float4*)(input1_shared + (((int)threadIdx.x) * 4)) = *(float4*)(input1 + ((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 8192));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 16384));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 24576));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 32768));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 640)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 40960));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49152));
            *(float4*)(input1_shared + ((((int)threadIdx.x) * 4) + 896)) = *(float4*)(input1 + (((((((((int)blockIdx.x) % 196) / 49) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 57344));
            __syncthreads();
            for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
              conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(rc_outer_inner * 8)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[(rc_outer_inner * 8)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((rc_outer_inner * 8) + 1)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[((rc_outer_inner * 8) + 1)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((rc_outer_inner * 8) + 64)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[((rc_outer_inner * 8) + 64)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((rc_outer_inner * 8) + 65)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[((rc_outer_inner * 8) + 65)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 8) + 4)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[((rc_outer_inner * 8) + 4)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((rc_outer_inner * 8) + 5)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[((rc_outer_inner * 8) + 5)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((rc_outer_inner * 8) + 68)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[((rc_outer_inner * 8) + 68)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((rc_outer_inner * 8) + 69)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[((rc_outer_inner * 8) + 69)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((rc_outer_inner * 8) + 2)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[((rc_outer_inner * 8) + 2)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((rc_outer_inner * 8) + 3)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[((rc_outer_inner * 8) + 3)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((rc_outer_inner * 8) + 66)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[((rc_outer_inner * 8) + 66)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((rc_outer_inner * 8) + 67)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[((rc_outer_inner * 8) + 67)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((rc_outer_inner * 8) + 6)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[((rc_outer_inner * 8) + 6)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((rc_outer_inner * 8) + 7)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[((rc_outer_inner * 8) + 7)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((rc_outer_inner * 8) + 70)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[((rc_outer_inner * 8) + 70)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((rc_outer_inner * 8) + 71)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[((rc_outer_inner * 8) + 71)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((rc_outer_inner * 8) + 128)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[((rc_outer_inner * 8) + 128)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((rc_outer_inner * 8) + 129)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[((rc_outer_inner * 8) + 129)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((rc_outer_inner * 8) + 192)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[28] = (conv2d_nchw[28] + (pad_temp_shared[((rc_outer_inner * 8) + 192)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((rc_outer_inner * 8) + 193)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[29] = (conv2d_nchw[29] + (pad_temp_shared[((rc_outer_inner * 8) + 193)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((rc_outer_inner * 8) + 132)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[((rc_outer_inner * 8) + 132)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((rc_outer_inner * 8) + 133)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[((rc_outer_inner * 8) + 133)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((rc_outer_inner * 8) + 196)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[28] = (conv2d_nchw[28] + (pad_temp_shared[((rc_outer_inner * 8) + 196)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((rc_outer_inner * 8) + 197)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[29] = (conv2d_nchw[29] + (pad_temp_shared[((rc_outer_inner * 8) + 197)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((rc_outer_inner * 8) + 130)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[((rc_outer_inner * 8) + 130)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((rc_outer_inner * 8) + 131)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[((rc_outer_inner * 8) + 131)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((rc_outer_inner * 8) + 194)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[30] = (conv2d_nchw[30] + (pad_temp_shared[((rc_outer_inner * 8) + 194)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[((rc_outer_inner * 8) + 195)] * input1_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 2))]));
              conv2d_nchw[31] = (conv2d_nchw[31] + (pad_temp_shared[((rc_outer_inner * 8) + 195)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 512)]));
              conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((rc_outer_inner * 8) + 134)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[((rc_outer_inner * 8) + 134)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((rc_outer_inner * 8) + 135)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[((rc_outer_inner * 8) + 135)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((rc_outer_inner * 8) + 198)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[30] = (conv2d_nchw[30] + (pad_temp_shared[((rc_outer_inner * 8) + 198)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
              conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[((rc_outer_inner * 8) + 199)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 1)]));
              conv2d_nchw[31] = (conv2d_nchw[31] + (pad_temp_shared[((rc_outer_inner * 8) + 199)] * input1_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 2)) + 513)]));
            }
          }
          for (int i0_inner = 0; i0_inner < 4; ++i0_inner) {
            for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
              for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
                compute[(((((((((((int)blockIdx.x) / 196) * 200704) + (i0_inner * 50176)) + (((((int)blockIdx.x) % 196) / 49) * 12544)) + (((int)threadIdx.x) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (i2_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + i3_inner)] = max((conv2d_nchw[(((i0_inner * 4) + (i2_inner * 2)) + i3_inner)] + input2[(((((((((((int)blockIdx.x) / 196) * 200704) + (i0_inner * 50176)) + (((((int)blockIdx.x) % 196) / 49) * 12544)) + (((int)threadIdx.x) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (i2_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + i3_inner)]), 0.000000e+00f);
                compute[((((((((((((int)blockIdx.x) / 196) * 200704) + (i0_inner * 50176)) + (((((int)blockIdx.x) % 196) / 49) * 12544)) + (((int)threadIdx.x) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (i2_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + i3_inner) + 6272)] = max((conv2d_nchw[((((i0_inner * 4) + (i2_inner * 2)) + i3_inner) + 16)] + input2[((((((((((((int)blockIdx.x) / 196) * 200704) + (i0_inner * 50176)) + (((((int)blockIdx.x) % 196) / 49) * 12544)) + (((int)threadIdx.x) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (i2_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + i3_inner) + 6272)]), 0.000000e+00f);
              }
            }
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

