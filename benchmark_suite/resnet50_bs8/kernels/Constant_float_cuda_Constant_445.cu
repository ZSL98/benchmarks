#include "shared.h"
// Node name:	Constant_445
// Description:	Constant
// Input:
// Output:
//	- name: Constant_445_0	type: float	shape: Shape{256, 512, 1, 1}
void Constant_float_cuda_Constant_445(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_445_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_445_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[524288];
    bin_file.read(tmp_mem, 524288);
    hipMemcpyAsync(output0, tmp_mem, 524288, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

