#include "shared.h"
// Node name:	Constant_478
// Description:	Constant
// Input:
// Output:
//	- name: Constant_478_0	type: float	shape: Shape{256, 256, 3, 3}
void Constant_float_cuda_Constant_478(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_478_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_478_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2359296];
    bin_file.read(tmp_mem, 2359296);
    hipMemcpyAsync(output0, tmp_mem, 2359296, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

