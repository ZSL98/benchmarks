#include "hip/hip_runtime.h"
#include "shared.h"
#define MIN(a,b) ((a)>(b)?(b):(a))
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}

#if CUDA_VERSION < 9000
#define CREATE_SHFL_MASK(mask, predicate) mask = 0u;
#else
#define FULL_WARP_MASK 0xFFFFFFFF
#define CREATE_SHFL_MASK(mask, predicate) \
  mask = __ballot_sync(FULL_WARP_MASK, (predicate))
#endif

__forceinline__ __device__ float CudaShuffleDownSync(unsigned mask, float val,
                                                     int delta,
                                                     int width = 32) {
#if CUDA_VERSION < 9000
  return __shfl_down(val, delta, width);
#else
  return __shfl_down_sync(mask, val, delta, width);
#endif
}

__device__ static float reduceMax(float val, int tid, int blockSize, float* shm) {
  unsigned mask = 0u;
  CREATE_SHFL_MASK(mask, tid < blockSize);

  val = max(val, CudaShuffleDownSync(mask, val, 16));
  val = max(val, CudaShuffleDownSync(mask, val, 8));
  val = max(val, CudaShuffleDownSync(mask, val, 4));
  val = max(val, CudaShuffleDownSync(mask, val, 2));
  val = max(val, CudaShuffleDownSync(mask, val, 1));

  if (tid < warpSize) shm[tid] = 0.;
  __syncthreads();

  if (tid % warpSize == 0) shm[tid / warpSize] = val;
  __syncthreads();

  CREATE_SHFL_MASK(mask, tid < warpSize);

  if (tid < warpSize) {
    val = shm[tid];

    val = max(val, CudaShuffleDownSync(mask, val, 16));
    val = max(val, CudaShuffleDownSync(mask, val, 8));
    val = max(val, CudaShuffleDownSync(mask, val, 4));
    val = max(val, CudaShuffleDownSync(mask, val, 2));
    val = max(val, CudaShuffleDownSync(mask, val, 1));
  }

  return val;
}

__device__ static float reduceSum(float val, int tid, int blockSize, float* shm) {
  unsigned mask = 0u;
  CREATE_SHFL_MASK(mask, tid < blockSize);

  val += CudaShuffleDownSync(mask, val, 16);
  val += CudaShuffleDownSync(mask, val, 8);
  val += CudaShuffleDownSync(mask, val, 4);
  val += CudaShuffleDownSync(mask, val, 2);
  val += CudaShuffleDownSync(mask, val, 1);

  if (tid < warpSize) shm[tid] = 0.;
  __syncthreads();

  if (tid % warpSize == 0) shm[tid / warpSize] = val;

  __syncthreads();

  CREATE_SHFL_MASK(mask, tid < warpSize);

  if (tid < warpSize) {
    val = shm[tid];

    val += CudaShuffleDownSync(mask, val, 16);
    val += CudaShuffleDownSync(mask, val, 8);
    val += CudaShuffleDownSync(mask, val, 4);
    val += CudaShuffleDownSync(mask, val, 2);
    val += CudaShuffleDownSync(mask, val, 1);
  }

  return val;
}
// Node name:	Sum_311
// Description:	Sum
// Input:
//	- name: Relu_310_0	type: float	shape: Shape{8, 2048, 7, 7}
// Output:
//	- name: Sum_311_0	type: float	shape: Shape{8, 2048}
extern "C" __launch_bounds__(32) __global__ void Sum_float_float_cuda_Sum_311(float* input0, float* output0)
{

    int width = 49;
    int block_size = 32;
    const int warp_size = 32;
    __shared__ float shm[warp_size];

    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int data_idx_offset = block_idx * width;

    float val = 0.0;
    for (int tidx = thread_idx; tidx < width; tidx += block_size) {
        int data_idx = tidx + data_idx_offset;
        val += input0[data_idx];
    }
    val = reduceSum(val, thread_idx, block_size, shm);
    if (thread_idx == 0) output0[block_idx] = val;


}
extern void Sum_float_float_cuda_Sum_311_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* output0) {
    Sum_float_float_cuda_Sum_311<<<grids, blocks, mem, stream>>>(input0, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

