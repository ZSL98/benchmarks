#include "shared.h"
// Node name:	Constant_514
// Description:	Constant
// Input:
// Output:
//	- name: Constant_514_0	type: float	shape: Shape{512, 2048, 1, 1}
void Constant_float_cuda_Constant_514(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_514_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_514_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4194304];
    bin_file.read(tmp_mem, 4194304);
    hipMemcpyAsync(output0, tmp_mem, 4194304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

