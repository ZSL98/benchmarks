#include "hip/hip_runtime.h"
#include "shared.h"
#include "nnfusion_rt.h"
#ifndef __HALF_COMPARE_EX__
#define __HALF_COMPARE_EX__
inline __device__ half max(half x, half y) { return x > y ? x : y; }
inline __device__ half min(half x, half y) { return x < y ? x : y; }
#endif
char* group_0_CUDA_GPU0_allocator_memory_pool;
float* Broadcast_317_0;
float* Broadcast_506_0;
float* Broadcast_449_0;
float* Broadcast_410_0;
float* Broadcast_377_0;
float* Broadcast_374_0;
float* Pad_73_0;
float* Relu_77_0;
float* MaxPool_78_0;
float* BatchNormInference_83_0;
float* Broadcast_386_0;
float* Broadcast_383_0;
float* Broadcast_380_0;
float* Relu_85_0;
float* Relu_89_0;
float* BatchNormInference_92_0;
float* Relu_94_0;
float* Broadcast_395_0;
float* Broadcast_392_0;
float* Broadcast_389_0;
float* Relu_98_0;
float* Relu_102_0;
float* BatchNormInference_105_0;
float* Relu_107_0;
float* Broadcast_404_0;
float* Broadcast_401_0;
float* Broadcast_398_0;
float* Relu_111_0;
float* Relu_115_0;
float* BatchNormInference_118_0;
float* Relu_120_0;
float* Pad_122_0;
float* BatchNormInference_128_0;
float* Broadcast_416_0;
float* Broadcast_413_0;
float* Broadcast_407_0;
float* Relu_129_0;
float* Pad_131_0;
float* Relu_135_0;
float* BatchNormInference_138_0;
float* Relu_140_0;
float* Broadcast_425_0;
float* Broadcast_422_0;
float* Broadcast_419_0;
float* Relu_144_0;
float* Relu_148_0;
float* BatchNormInference_151_0;
float* Relu_153_0;
float* Broadcast_434_0;
float* Broadcast_431_0;
float* Broadcast_428_0;
float* Relu_157_0;
float* Relu_161_0;
float* BatchNormInference_164_0;
float* Relu_166_0;
float* Broadcast_443_0;
float* Broadcast_440_0;
float* Broadcast_437_0;
float* Relu_170_0;
float* Relu_174_0;
float* BatchNormInference_177_0;
float* Relu_179_0;
float* Pad_181_0;
float* BatchNormInference_187_0;
float* Broadcast_455_0;
float* Broadcast_452_0;
float* Broadcast_446_0;
float* Relu_188_0;
float* Pad_190_0;
float* Relu_194_0;
float* BatchNormInference_197_0;
float* Relu_199_0;
float* Broadcast_464_0;
float* Broadcast_461_0;
float* Broadcast_458_0;
float* Relu_203_0;
float* Relu_207_0;
float* BatchNormInference_210_0;
float* Relu_212_0;
float* Broadcast_473_0;
float* Broadcast_470_0;
float* Broadcast_467_0;
float* Relu_216_0;
float* Relu_220_0;
float* BatchNormInference_223_0;
float* Relu_225_0;
float* Broadcast_482_0;
float* Broadcast_479_0;
float* Broadcast_476_0;
float* Relu_229_0;
float* Relu_233_0;
float* BatchNormInference_236_0;
float* Relu_238_0;
float* Broadcast_491_0;
float* Broadcast_488_0;
float* Broadcast_485_0;
float* Relu_242_0;
float* Relu_246_0;
float* BatchNormInference_249_0;
float* Relu_251_0;
float* Broadcast_500_0;
float* Broadcast_497_0;
float* Broadcast_494_0;
float* Relu_255_0;
float* Relu_259_0;
float* BatchNormInference_262_0;
float* Relu_264_0;
float* Pad_268_0;
float* BatchNormInference_273_0;
float* Broadcast_512_0;
float* Broadcast_509_0;
float* Broadcast_503_0;
float* Relu_272_0;
float* Pad_275_0;
float* Relu_279_0;
float* BatchNormInference_282_0;
float* Relu_284_0;
float* Broadcast_521_0;
float* Broadcast_518_0;
float* Broadcast_515_0;
float* Relu_288_0;
float* Relu_292_0;
float* BatchNormInference_295_0;
float* Relu_297_0;
float* Broadcast_530_0;
float* Broadcast_527_0;
float* Broadcast_524_0;
float* Relu_301_0;
float* Relu_305_0;
float* BatchNormInference_308_0;
float* Relu_310_0;
float* Sum_311_0;
float* Divide_313_0;
float* Reshape_314_0;
float* Reshape_315_0;
float* Dot_316_0;
float* Add_318_0;
float* Result_319_0;
char* group_persist_CUDA_GPU0_allocator_memory_pool;
float* Constant_71_0;
float* Constant_70_0;
float* Constant_312_0;
float* Constant_54_0;
float* Constant_505_0;
float* Constant_267_0;
float* Constant_23_0;
float* Constant_448_0;
float* Constant_180_0;
float* Constant_49_0;
float* Constant_409_0;
float* Constant_121_0;
float* Constant_2_0;
float* Constant_376_0;
float* Constant_10_0;
float* Constant_373_0;
float* Constant_72_0;
float* Constant_385_0;
float* Constant_382_0;
float* Constant_379_0;
float* Constant_394_0;
float* Constant_391_0;
float* Constant_388_0;
float* Constant_403_0;
float* Constant_400_0;
float* Constant_397_0;
float* Constant_415_0;
float* Constant_35_0;
float* Constant_412_0;
float* Constant_130_0;
float* Constant_406_0;
float* Constant_424_0;
float* Constant_421_0;
float* Constant_418_0;
float* Constant_433_0;
float* Constant_430_0;
float* Constant_427_0;
float* Constant_442_0;
float* Constant_439_0;
float* Constant_436_0;
float* Constant_454_0;
float* Constant_451_0;
float* Constant_189_0;
float* Constant_445_0;
float* Constant_463_0;
float* Constant_460_0;
float* Constant_457_0;
float* Constant_472_0;
float* Constant_469_0;
float* Constant_466_0;
float* Constant_481_0;
float* Constant_478_0;
float* Constant_475_0;
float* Constant_490_0;
float* Constant_487_0;
float* Constant_484_0;
float* Constant_499_0;
float* Constant_496_0;
float* Constant_493_0;
float* Constant_511_0;
float* Constant_508_0;
float* Constant_274_0;
float* Constant_502_0;
float* Constant_520_0;
float* Constant_517_0;
float* Constant_514_0;
float* Constant_529_0;
float* Constant_526_0;
float* Constant_523_0;

extern void Constant_float_cuda_Constant_469(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_448(hipStream_t stream, float* output0);

extern void Pad_float_float_float_cuda_Pad_73_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Constant_float_cuda_Constant_180(hipStream_t stream, float* output0);

extern void MaxPool_float_float_cuda_MaxPool_78_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_562_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_532_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Sum_float_float_cuda_Sum_311_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* output0);

extern void Dot_float_float_float_cuda_Dot_316_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* __restrict__ input0,  float* __restrict__ input1,  float* __restrict__ output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_542_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Result_float_float_cuda_lib_Result_319(float* input0, float** output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_560_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_582_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Pad_float_float_float_cuda_Pad_275_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_539_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_535_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Add_float_float_float_cuda_Add_318_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_569_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_534_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_445(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_533_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Pad_float_float_float_cuda_Pad_131_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Pad_float_float_float_cuda_Pad_268_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Pad_float_float_float_cuda_Pad_181_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Pad_float_float_float_cuda_Pad_122_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Pad_float_float_float_cuda_Pad_190_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_548_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_559_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_581_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_457(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_49(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_400(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_397(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_487(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_23(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_442(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_496(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_451(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_541_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_424(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_466(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_499(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_520(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_130(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_376(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_35(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_561_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_523(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_547_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_511(hipStream_t stream, float* output0);

extern void Broadcast_float_float_cuda_Broadcast_317_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* output0);

extern void Constant_float_cuda_Constant_415(hipStream_t stream, float* output0);

extern void FusedKernel_float_float_float_cuda_Add_Relu_0_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Constant_float_cuda_Constant_121(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_379(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_385(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_70(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_436(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_421(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_412(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_391(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_490(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_312(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_10(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_430(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_570_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

#ifndef __NNFUSION_GRAPH_CONFIG__
#define __NNFUSION_GRAPH_CONFIG__
#define NNFUSION_GRAPH_INPUT_NUM 1
#define NNFUSION_GRAPH_OUTPUT_NUM 1
#define NNFUSION_GRAPH_INPUT_DTYPE_0 float
#define NNFUSION_GRAPH_INPUT_SHAPE_0 {8, 3, 224, 224}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_0 float
#define NNFUSION_GRAPH_OUTPUT_SHAPE_0 {8, 1000}
#endif


extern void Constant_float_cuda_Constant_481(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_439(hipStream_t stream, float* output0);

extern void Divide_float_float_float_cuda_Divide_313_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0);

extern void Constant_float_cuda_Constant_394(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_529(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_409(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_189(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_403(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_475(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_517(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_540_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_267(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_406(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_514(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_550_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_71(hipStream_t stream, float* output0);

extern void Broadcast_float_float_cuda_lib_Broadcast_506(float* input0, float* output0);

extern void Constant_float_cuda_Constant_478(hipStream_t stream, float* output0);
// 0: CUDA_GPU; 1: ROCM_GPU; 2: GENERIC_CPU; 3: HLSL; 4: GraphCore; 5: UNKNOWN
int get_device_type()
{
    return 0;
}

extern void Constant_float_cuda_Constant_373(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_382(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_565_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_274(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_388(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_427(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_472(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_454(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_463(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_418(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_460(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_2(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_484(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_526(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_502(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_508(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_433(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_72(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_505(hipStream_t stream, float* output0);

extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_574_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0);

extern void Constant_float_cuda_Constant_54(hipStream_t stream, float* output0);

extern void Constant_float_cuda_Constant_493(hipStream_t stream, float* output0);

extern "C" void cuda_init()
{
CUDA_SAFE_CALL(hipDeviceReset());
// total memory:214524032
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipMalloc((void**)&group_0_CUDA_GPU0_allocator_memory_pool,112426240));
CUDA_SAFE_CALL(hipMemset((void*)group_0_CUDA_GPU0_allocator_memory_pool, 0, 112426240));
Broadcast_317_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+0);
Broadcast_506_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Broadcast_449_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_410_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_377_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_374_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
Pad_73_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+73891072);
Relu_77_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+78969472);
MaxPool_78_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
BatchNormInference_83_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+54623488);
Broadcast_386_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_383_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+80313600);
Broadcast_380_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+86736128);
Relu_85_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+93158656);
Relu_89_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
BatchNormInference_92_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+80313600);
Relu_94_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_395_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
Broadcast_392_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+73891072);
Broadcast_389_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+80313600);
Relu_98_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+86736128);
Relu_102_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+80313600);
BatchNormInference_105_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+86736128);
Relu_107_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
Broadcast_404_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_401_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+73891072);
Broadcast_398_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+80313600);
Relu_111_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+86736128);
Relu_115_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+80313600);
BatchNormInference_118_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+86736128);
Relu_120_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Pad_122_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
BatchNormInference_128_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+73891072);
Broadcast_416_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_413_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+48200960);
Broadcast_407_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+51412224);
Relu_129_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+86736128);
Pad_131_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Relu_135_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+36289792);
BatchNormInference_138_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Relu_140_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_425_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_422_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+35355904);
Broadcast_419_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+38567168);
Relu_144_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Relu_148_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+38567168);
BatchNormInference_151_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Relu_153_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_434_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_431_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+35355904);
Broadcast_428_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+38567168);
Relu_157_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Relu_161_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+38567168);
BatchNormInference_164_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Relu_166_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_443_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Broadcast_440_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+35355904);
Broadcast_437_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+38567168);
Relu_170_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Relu_174_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+38567168);
BatchNormInference_177_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Relu_179_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+22510848);
Pad_181_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
BatchNormInference_187_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+35355904);
Broadcast_455_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_452_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_446_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+11271424);
Relu_188_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+41778432);
Pad_190_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+11271424);
Relu_194_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+18644224);
BatchNormInference_197_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Relu_199_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_464_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_461_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+16088320);
Broadcast_458_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
Relu_203_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_207_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
BatchNormInference_210_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_212_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_473_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_470_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+16088320);
Broadcast_467_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
Relu_216_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_220_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
BatchNormInference_223_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_225_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_482_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_479_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+16088320);
Broadcast_476_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
Relu_229_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_233_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
BatchNormInference_236_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_238_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_491_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_488_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+16088320);
Broadcast_485_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
Relu_242_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_246_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
BatchNormInference_249_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_251_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_500_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Broadcast_497_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+16088320);
Broadcast_494_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
Relu_255_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_259_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+17693952);
BatchNormInference_262_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Relu_264_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+9665792);
Pad_268_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
BatchNormInference_273_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+16088320);
Broadcast_512_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Broadcast_509_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_503_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+4046080);
Relu_272_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+19299584);
Pad_275_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+4046080);
Relu_279_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+8240384);
BatchNormInference_282_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Relu_284_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Broadcast_521_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_518_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+6454528);
Broadcast_515_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+7257344);
Relu_288_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+8060160);
Relu_292_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+7257344);
BatchNormInference_295_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+8060160);
Relu_297_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Broadcast_530_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Broadcast_527_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+6454528);
Broadcast_524_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+7257344);
Relu_301_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+8060160);
Relu_305_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+7257344);
BatchNormInference_308_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+8060160);
Relu_310_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Sum_311_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Divide_313_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Reshape_314_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Reshape_315_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3243264);
Dot_316_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Add_318_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
Result_319_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+32000);
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipMalloc((void**)&group_persist_CUDA_GPU0_allocator_memory_pool,102097792));
CUDA_SAFE_CALL(hipMemset((void*)group_persist_CUDA_GPU0_allocator_memory_pool, 0, 102097792));
Constant_71_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+0);
Constant_70_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+4032);
Constant_312_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+8196032);
Constant_54_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+8261568);
Constant_505_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+8269760);
Constant_267_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+16658368);
Constant_23_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+16658432);
Constant_448_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+16662528);
Constant_180_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+18759680);
Constant_49_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+18759744);
Constant_409_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+18761792);
Constant_121_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19286080);
Constant_2_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19286144);
Constant_376_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19287168);
Constant_10_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19352704);
Constant_373_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19352960);
Constant_72_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19390592);
Constant_385_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19390656);
Constant_382_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19456192);
Constant_379_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19603648);
Constant_394_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19620032);
Constant_391_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19685568);
Constant_388_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19833024);
Constant_403_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19898560);
Constant_400_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+19964096);
Constant_397_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+20111552);
Constant_415_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+20177088);
Constant_35_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+20439232);
Constant_412_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+20439744);
Constant_130_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+21029568);
Constant_406_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+21029632);
Constant_424_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+21160704);
Constant_421_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+21422848);
Constant_418_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+22012672);
Constant_433_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+22274816);
Constant_430_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+22536960);
Constant_427_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+23126784);
Constant_442_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+23388928);
Constant_439_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+23651072);
Constant_436_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+24240896);
Constant_454_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+24503040);
Constant_451_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+25551616);
Constant_189_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+27910912);
Constant_445_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+27910976);
Constant_463_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+28435264);
Constant_460_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+29483840);
Constant_457_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+31843136);
Constant_472_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+32891712);
Constant_469_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+33940288);
Constant_466_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+36299584);
Constant_481_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+37348160);
Constant_478_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+38396736);
Constant_475_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+40756032);
Constant_490_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+41804608);
Constant_487_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+42853184);
Constant_484_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+45212480);
Constant_499_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+46261056);
Constant_496_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+47309632);
Constant_493_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+49668928);
Constant_511_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+50717504);
Constant_508_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+54911808);
Constant_274_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+64348992);
Constant_502_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+64349056);
Constant_520_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+66446208);
Constant_517_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+70640512);
Constant_514_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+80077696);
Constant_529_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+84272000);
Constant_526_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+88466304);
Constant_523_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+97903488);
// create streams/handles
 // name=resnet_model/dense/bias/read/_265__cf__265
//Const load, Constant_71_0);
 // name=resnet_model/dense/kernel/read/_266__cf__266
//Const load, Constant_70_0);
 // name=Constant_312
//Const load, Constant_312_0);
 // name=resnet_model/batch_normalization_43/beta/read/_152__cf__152
//Const load, Constant_54_0);
 // name=Constant_505
//Const load, Constant_505_0);
 // name=Constant_267
//Const load, Constant_267_0);
 // name=resnet_model/batch_normalization_24/beta/read/_68__cf__68
//Const load, Constant_23_0);
 // name=Constant_448
//Const load, Constant_448_0);
 // name=Constant_180
//Const load, Constant_180_0);
 // name=resnet_model/batch_normalization_11/beta/read/_12__cf__12
//Const load, Constant_49_0);
 // name=Constant_409
//Const load, Constant_409_0);
 // name=Constant_121
//Const load, Constant_121_0);
 // name=resnet_model/batch_normalization_1/beta/read/_4__cf__4
//Const load, Constant_2_0);
 // name=Constant_376
//Const load, Constant_376_0);
 // name=resnet_model/batch_normalization/beta/read/_0__cf__0
//Const load, Constant_10_0);
 // name=Constant_373
//Const load, Constant_373_0);
 // name=Constant_72
//Const load, Constant_72_0);
 // name=Constant_385
//Const load, Constant_385_0);
 // name=Constant_382
//Const load, Constant_382_0);
 // name=Constant_379
//Const load, Constant_379_0);
 // name=Constant_394
//Const load, Constant_394_0);
 // name=Constant_391
//Const load, Constant_391_0);
 // name=Constant_388
//Const load, Constant_388_0);
 // name=Constant_403
//Const load, Constant_403_0);
 // name=Constant_400
//Const load, Constant_400_0);
 // name=Constant_397
//Const load, Constant_397_0);
 // name=Constant_415
//Const load, Constant_415_0);
 // name=resnet_model/batch_normalization_12/beta/read/_16__cf__16
//Const load, Constant_35_0);
 // name=Constant_412
//Const load, Constant_412_0);
 // name=Constant_130
//Const load, Constant_130_0);
 // name=Constant_406
//Const load, Constant_406_0);
 // name=Constant_424
//Const load, Constant_424_0);
 // name=Constant_421
//Const load, Constant_421_0);
 // name=Constant_418
//Const load, Constant_418_0);
 // name=Constant_433
//Const load, Constant_433_0);
 // name=Constant_430
//Const load, Constant_430_0);
 // name=Constant_427
//Const load, Constant_427_0);
 // name=Constant_442
//Const load, Constant_442_0);
 // name=Constant_439
//Const load, Constant_439_0);
 // name=Constant_436
//Const load, Constant_436_0);
 // name=Constant_454
//Const load, Constant_454_0);
 // name=Constant_451
//Const load, Constant_451_0);
 // name=Constant_189
//Const load, Constant_189_0);
 // name=Constant_445
//Const load, Constant_445_0);
 // name=Constant_463
//Const load, Constant_463_0);
 // name=Constant_460
//Const load, Constant_460_0);
 // name=Constant_457
//Const load, Constant_457_0);
 // name=Constant_472
//Const load, Constant_472_0);
 // name=Constant_469
//Const load, Constant_469_0);
 // name=Constant_466
//Const load, Constant_466_0);
 // name=Constant_481
//Const load, Constant_481_0);
 // name=Constant_478
//Const load, Constant_478_0);
 // name=Constant_475
//Const load, Constant_475_0);
 // name=Constant_490
//Const load, Constant_490_0);
 // name=Constant_487
//Const load, Constant_487_0);
 // name=Constant_484
//Const load, Constant_484_0);
 // name=Constant_499
//Const load, Constant_499_0);
 // name=Constant_496
//Const load, Constant_496_0);
 // name=Constant_493
//Const load, Constant_493_0);
 // name=Constant_511
//Const load, Constant_511_0);
 // name=Constant_508
//Const load, Constant_508_0);
 // name=Constant_274
//Const load, Constant_274_0);
 // name=Constant_502
//Const load, Constant_502_0);
 // name=Constant_520
//Const load, Constant_520_0);
 // name=Constant_517
//Const load, Constant_517_0);
 // name=Constant_514
//Const load, Constant_514_0);
 // name=Constant_529
//Const load, Constant_529_0);
 // name=Constant_526
//Const load, Constant_526_0);
 // name=Constant_523
//Const load, Constant_523_0);
CUDA_SAFE_CALL(hipDeviceGetAttribute(&num_SMs, hipDeviceAttributeMultiprocessorCount, 0));
}


extern "C" int kernel_entry(float* Parameter_30_0, float** Result_319_0)
{
// kernel_entry_init
 // name=Broadcast_317
Broadcast_float_float_cuda_Broadcast_317_Call(dim3(125, 1, 1), dim3(64, 1, 1), 0, 0, Constant_71_0, Broadcast_317_0);
 // name=Broadcast_506
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_54_0, Broadcast_506_0);
 // name=Broadcast_449
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_449_0);
 // name=Broadcast_410
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_410_0);
 // name=Broadcast_377
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_377_0);
 // name=Broadcast_374
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_374_0);
 // name=resnet_model/Pad
Pad_float_float_float_cuda_Pad_73_Call(dim3(19838, 1, 1), dim3(64, 1, 1), 0, 0, Parameter_30_0, Constant_72_0, Pad_73_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_532_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Pad_73_0, Constant_373_0, Broadcast_374_0, Relu_77_0);
 // name=resnet_model/max_pooling2d/MaxPool
// MaxPool_float_float_cuda_MaxPool_78_Call(dim3(28160, 1, 1), dim3(55, 1, 1), 0, 0, Relu_77_0, MaxPool_78_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_565_Call(dim3(1792, 1, 1), dim3(224, 1, 1), 0, 0, MaxPool_78_0, Constant_376_0, Broadcast_377_0, BatchNormInference_83_0);
 // name=Broadcast_386
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_386_0);
 // name=Broadcast_383
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_383_0);
 // name=Broadcast_380
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_380_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_533_Call(dim3(448, 1, 1), dim3(224, 1, 1), 0, 0, MaxPool_78_0, Constant_379_0, Broadcast_380_0, Relu_85_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_534_Call(dim3(98, 1, 1), dim3(256, 1, 1), 0, 0, Relu_85_0, Constant_382_0, Broadcast_383_0, Relu_89_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_565_Call(dim3(1792, 1, 1), dim3(224, 1, 1), 0, 0, Relu_89_0, Constant_385_0, Broadcast_386_0, BatchNormInference_92_0);
 // name=ElementWiseFused_585
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(25088, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_92_0, BatchNormInference_83_0, Relu_94_0);
 // name=Broadcast_395
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_395_0);
 // name=Broadcast_392
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_392_0);
 // name=Broadcast_389
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_389_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_535_Call(dim3(392, 1, 1), dim3(256, 1, 1), 0, 0, Relu_94_0, Constant_388_0, Broadcast_389_0, Relu_98_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_534_Call(dim3(98, 1, 1), dim3(256, 1, 1), 0, 0, Relu_98_0, Constant_391_0, Broadcast_392_0, Relu_102_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_565_Call(dim3(1792, 1, 1), dim3(224, 1, 1), 0, 0, Relu_102_0, Constant_394_0, Broadcast_395_0, BatchNormInference_105_0);
 // name=ElementWiseFused_586
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(25088, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_105_0, Relu_94_0, Relu_107_0);
 // name=Broadcast_404
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_404_0);
 // name=Broadcast_401
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_401_0);
 // name=Broadcast_398
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_10_0, Broadcast_398_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_535_Call(dim3(392, 1, 1), dim3(256, 1, 1), 0, 0, Relu_107_0, Constant_397_0, Broadcast_398_0, Relu_111_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_534_Call(dim3(98, 1, 1), dim3(256, 1, 1), 0, 0, Relu_111_0, Constant_400_0, Broadcast_401_0, Relu_115_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_565_Call(dim3(1792, 1, 1), dim3(224, 1, 1), 0, 0, Relu_115_0, Constant_403_0, Broadcast_404_0, BatchNormInference_118_0);
 // name=ElementWiseFused_587
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(25088, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_118_0, Relu_107_0, Relu_120_0);
 // name=resnet_model/Pad_1
Pad_float_float_float_cuda_Pad_122_Call(dim3(100352, 1, 1), dim3(64, 1, 1), 0, 0, Relu_120_0, Constant_121_0, Pad_122_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_569_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Pad_122_0, Constant_409_0, Broadcast_410_0, BatchNormInference_128_0);
 // name=Broadcast_416
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_416_0);
 // name=Broadcast_413
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_413_0);
 // name=Broadcast_407
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_407_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_539_Call(dim3(784, 1, 1), dim3(256, 1, 1), 0, 0, Relu_120_0, Constant_406_0, Broadcast_407_0, Relu_129_0);
 // name=resnet_model/Pad_2
Pad_float_float_float_cuda_Pad_131_Call(dim3(53824, 1, 1), dim3(64, 1, 1), 0, 0, Relu_129_0, Constant_130_0, Pad_131_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_540_Call(dim3(392, 1, 1), dim3(128, 1, 1), 0, 0, Pad_131_0, Constant_412_0, Broadcast_413_0, Relu_135_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_570_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_135_0, Constant_415_0, Broadcast_416_0, BatchNormInference_138_0);
 // name=ElementWiseFused_588
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(12544, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_138_0, BatchNormInference_128_0, Relu_140_0);
 // name=Broadcast_425
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_425_0);
 // name=Broadcast_422
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_422_0);
 // name=Broadcast_419
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_419_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_541_Call(dim3(224, 1, 1), dim3(224, 1, 1), 0, 0, Relu_140_0, Constant_418_0, Broadcast_419_0, Relu_144_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_542_Call(dim3(196, 1, 1), dim3(128, 1, 1), 0, 0, Relu_144_0, Constant_421_0, Broadcast_422_0, Relu_148_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_570_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_148_0, Constant_424_0, Broadcast_425_0, BatchNormInference_151_0);
 // name=ElementWiseFused_589
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(12544, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_151_0, Relu_140_0, Relu_153_0);
 // name=Broadcast_434
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_434_0);
 // name=Broadcast_431
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_431_0);
 // name=Broadcast_428
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_428_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_541_Call(dim3(224, 1, 1), dim3(224, 1, 1), 0, 0, Relu_153_0, Constant_427_0, Broadcast_428_0, Relu_157_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_542_Call(dim3(196, 1, 1), dim3(128, 1, 1), 0, 0, Relu_157_0, Constant_430_0, Broadcast_431_0, Relu_161_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_570_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_161_0, Constant_433_0, Broadcast_434_0, BatchNormInference_164_0);
 // name=ElementWiseFused_590
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(12544, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_164_0, Relu_153_0, Relu_166_0);
 // name=Broadcast_443
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_443_0);
 // name=Broadcast_440
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_440_0);
 // name=Broadcast_437
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_35_0, Broadcast_437_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_541_Call(dim3(224, 1, 1), dim3(224, 1, 1), 0, 0, Relu_166_0, Constant_436_0, Broadcast_437_0, Relu_170_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_542_Call(dim3(196, 1, 1), dim3(128, 1, 1), 0, 0, Relu_170_0, Constant_439_0, Broadcast_440_0, Relu_174_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_570_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_174_0, Constant_442_0, Broadcast_443_0, BatchNormInference_177_0);
 // name=ElementWiseFused_591
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(12544, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_177_0, Relu_166_0, Relu_179_0);
 // name=resnet_model/Pad_3
Pad_float_float_float_cuda_Pad_181_Call(dim3(50176, 1, 1), dim3(64, 1, 1), 0, 0, Relu_179_0, Constant_180_0, Pad_181_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_574_Call(dim3(784, 1, 1), dim3(32, 1, 1), 0, 0, Pad_181_0, Constant_448_0, Broadcast_449_0, BatchNormInference_187_0);
 // name=Broadcast_455
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_455_0);
 // name=Broadcast_452
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_452_0);
 // name=Broadcast_446
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_446_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_547_Call(dim3(448, 1, 1), dim3(224, 1, 1), 0, 0, Relu_179_0, Constant_445_0, Broadcast_446_0, Relu_188_0);
 // name=resnet_model/Pad_4
Pad_float_float_float_cuda_Pad_190_Call(dim3(28800, 1, 1), dim3(64, 1, 1), 0, 0, Relu_188_0, Constant_189_0, Pad_190_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_548_Call(dim3(196, 1, 1), dim3(128, 1, 1), 0, 0, Pad_190_0, Constant_451_0, Broadcast_452_0, Relu_194_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_194_0, Constant_454_0, Broadcast_455_0, BatchNormInference_197_0);
 // name=ElementWiseFused_592
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(6272, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_197_0, BatchNormInference_187_0, Relu_199_0);
 // name=Broadcast_464
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_464_0);
 // name=Broadcast_461
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_461_0);
 // name=Broadcast_458
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_458_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(dim3(392, 1, 1), dim3(32, 1, 1), 0, 0, Relu_199_0, Constant_457_0, Broadcast_458_0, Relu_203_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_550_Call(dim3(448, 1, 1), dim3(112, 1, 1), 0, 0, Relu_203_0, Constant_460_0, Broadcast_461_0, Relu_207_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_207_0, Constant_463_0, Broadcast_464_0, BatchNormInference_210_0);
 // name=ElementWiseFused_593
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(6272, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_210_0, Relu_199_0, Relu_212_0);
 // name=Broadcast_473
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_473_0);
 // name=Broadcast_470
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_470_0);
 // name=Broadcast_467
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_467_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(dim3(392, 1, 1), dim3(32, 1, 1), 0, 0, Relu_212_0, Constant_466_0, Broadcast_467_0, Relu_216_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_550_Call(dim3(448, 1, 1), dim3(112, 1, 1), 0, 0, Relu_216_0, Constant_469_0, Broadcast_470_0, Relu_220_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_220_0, Constant_472_0, Broadcast_473_0, BatchNormInference_223_0);
 // name=ElementWiseFused_594
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(6272, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_223_0, Relu_212_0, Relu_225_0);
 // name=Broadcast_482
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_482_0);
 // name=Broadcast_479
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_479_0);
 // name=Broadcast_476
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_476_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(dim3(392, 1, 1), dim3(32, 1, 1), 0, 0, Relu_225_0, Constant_475_0, Broadcast_476_0, Relu_229_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_550_Call(dim3(448, 1, 1), dim3(112, 1, 1), 0, 0, Relu_229_0, Constant_478_0, Broadcast_479_0, Relu_233_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_233_0, Constant_481_0, Broadcast_482_0, BatchNormInference_236_0);
 // name=ElementWiseFused_595
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(6272, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_236_0, Relu_225_0, Relu_238_0);
 // name=Broadcast_491
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_491_0);
 // name=Broadcast_488
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_488_0);
 // name=Broadcast_485
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_485_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(dim3(392, 1, 1), dim3(32, 1, 1), 0, 0, Relu_238_0, Constant_484_0, Broadcast_485_0, Relu_242_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_550_Call(dim3(448, 1, 1), dim3(112, 1, 1), 0, 0, Relu_242_0, Constant_487_0, Broadcast_488_0, Relu_246_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_246_0, Constant_490_0, Broadcast_491_0, BatchNormInference_249_0);
 // name=ElementWiseFused_596
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(6272, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_249_0, Relu_238_0, Relu_251_0);
 // name=Broadcast_500
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_23_0, Broadcast_500_0);
 // name=Broadcast_497
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_497_0);
 // name=Broadcast_494
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_2_0, Broadcast_494_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_549_Call(dim3(392, 1, 1), dim3(32, 1, 1), 0, 0, Relu_251_0, Constant_493_0, Broadcast_494_0, Relu_255_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_550_Call(dim3(448, 1, 1), dim3(112, 1, 1), 0, 0, Relu_255_0, Constant_496_0, Broadcast_497_0, Relu_259_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(dim3(896, 1, 1), dim3(224, 1, 1), 0, 0, Relu_259_0, Constant_499_0, Broadcast_500_0, BatchNormInference_262_0);
 // name=ElementWiseFused_597
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(6272, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_262_0, Relu_251_0, Relu_264_0);
 // name=resnet_model/Pad_5
Pad_float_float_float_cuda_Pad_268_Call(dim3(25088, 1, 1), dim3(64, 1, 1), 0, 0, Relu_264_0, Constant_267_0, Pad_268_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_581_Call(dim3(224, 1, 1), dim3(224, 1, 1), 0, 0, Pad_268_0, Constant_505_0, Broadcast_506_0, BatchNormInference_273_0);
 // name=Broadcast_512
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_54_0, Broadcast_512_0);
 // name=Broadcast_509
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_509_0);
 // name=Broadcast_503
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_503_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_559_Call(dim3(448, 1, 1), dim3(56, 1, 1), 0, 0, Relu_264_0, Constant_502_0, Broadcast_503_0, Relu_272_0);
 // name=resnet_model/Pad_6
Pad_float_float_float_cuda_Pad_275_Call(dim3(16384, 1, 1), dim3(64, 1, 1), 0, 0, Relu_272_0, Constant_274_0, Pad_275_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_560_Call(dim3(224, 1, 1), dim3(56, 1, 1), 0, 0, Pad_275_0, Constant_508_0, Broadcast_509_0, Relu_279_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_582_Call(dim3(512, 1, 1), dim3(56, 1, 1), 0, 0, Relu_279_0, Constant_511_0, Broadcast_512_0, BatchNormInference_282_0);
 // name=ElementWiseFused_598
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(3136, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_282_0, BatchNormInference_273_0, Relu_284_0);
 // name=Broadcast_521
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_54_0, Broadcast_521_0);
 // name=Broadcast_518
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_518_0);
 // name=Broadcast_515
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_515_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_561_Call(dim3(224, 1, 1), dim3(64, 1, 1), 0, 0, Relu_284_0, Constant_514_0, Broadcast_515_0, Relu_288_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_562_Call(dim3(256, 1, 1), dim3(112, 1, 1), 0, 0, Relu_288_0, Constant_517_0, Broadcast_518_0, Relu_292_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_582_Call(dim3(512, 1, 1), dim3(56, 1, 1), 0, 0, Relu_292_0, Constant_520_0, Broadcast_521_0, BatchNormInference_295_0);
 // name=ElementWiseFused_599
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(3136, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_295_0, Relu_284_0, Relu_297_0);
 // name=Broadcast_530
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_54_0, Broadcast_530_0);
 // name=Broadcast_527
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_527_0);
 // name=Broadcast_524
Broadcast_float_float_cuda_lib_Broadcast_506(Constant_49_0, Broadcast_524_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_561_Call(dim3(224, 1, 1), dim3(64, 1, 1), 0, 0, Relu_297_0, Constant_523_0, Broadcast_524_0, Relu_301_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_562_Call(dim3(256, 1, 1), dim3(112, 1, 1), 0, 0, Relu_301_0, Constant_526_0, Broadcast_527_0, Relu_305_0);
 // name=Matched_Pattern
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_582_Call(dim3(512, 1, 1), dim3(56, 1, 1), 0, 0, Relu_305_0, Constant_529_0, Broadcast_530_0, BatchNormInference_308_0);
 // name=ElementWiseFused_600
FusedKernel_float_float_float_cuda_Add_Relu_0_Call(dim3(3136, 1, 1), dim3(128, 1, 1), 0, 0, BatchNormInference_308_0, Relu_297_0, Relu_310_0);
 // name=Sum_311
Sum_float_float_cuda_Sum_311_Call(dim3(16384, 1, 1), dim3(32, 1, 1), 0, 0, Relu_310_0, Sum_311_0);
 // name=Divide_313
Divide_float_float_float_cuda_Divide_313_Call(dim3(64, 1, 1), dim3(128, 1, 1), 0, 0, Sum_311_0, Constant_312_0, Divide_313_0);
 // name=resnet_model/Mean
// eliminated: Reshape_float_float_cuda_lib_Reshape_314(0, Divide_313_0, Reshape_314_0);
 // name=resnet_model/Squeeze
// eliminated: Reshape_float_float_cuda_lib_Reshape_315(0, Reshape_314_0, Reshape_315_0);
 // name=resnet_model/dense/MatMul
// Dot_float_float_float_cuda_Dot_316_Call(dim3(100, 1, 1), dim3(80, 1, 1), 0, 0, Reshape_315_0, Constant_70_0, Dot_316_0);
 // name=resnet_model/dense/BiasAdd
Add_float_float_float_cuda_Add_318_Call(dim3(62, 1, 1), dim3(64, 1, 1), 0, 0, Dot_316_0, Broadcast_317_0, Add_318_0);
 // name=Result_319
Result_float_float_cuda_lib_Result_319(Add_318_0, Result_319_0);
return 0;
}


extern "C" void cuda_free()
{
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipFree(group_0_CUDA_GPU0_allocator_memory_pool));
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipFree(group_persist_CUDA_GPU0_allocator_memory_pool));
}

// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

