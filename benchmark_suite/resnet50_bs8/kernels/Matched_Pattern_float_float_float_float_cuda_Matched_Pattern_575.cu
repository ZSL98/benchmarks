#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_575
// Description:	Matched_Pattern
// Input:
//	- name: Relu_194_0	type: float	shape: Shape{8, 256, 14, 14}
//	- name: Constant_454_0	type: float	shape: Shape{1024, 256, 1, 1}
//	- name: Broadcast_455_0	type: float	shape: Shape{8, 1024, 14, 14}
// Output:
//	- name: BatchNormInference_197_0	type: float	shape: Shape{8, 1024, 14, 14}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[224];
    __shared__ float input1_shared[512];
    {
        float* T_add = output0;
        {
          float conv2d_nchw[8];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
            __syncthreads();
            pad_temp_shared[((int)threadIdx.x)] = input0[((((((((int)blockIdx.x) / 112) * 50176) + (rc_outer_outer * 1568)) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28))];
            input1_shared[((int)threadIdx.x)] = input1[((((((((int)blockIdx.x) % 112) / 7) * 16384) + ((((int)threadIdx.x) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
            input1_shared[(((int)threadIdx.x) + 224)] = input1[(((((((((int)blockIdx.x) % 112) / 7) * 16384) + ((((int)threadIdx.x) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 7168)];
            if (((int)threadIdx.x) < 64) {
              input1_shared[(((int)threadIdx.x) + 448)] = input1[(((((((((int)blockIdx.x) % 112) / 7) * 16384) + ((((int)threadIdx.x) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 14336)];
            }
            __syncthreads();
            for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
              for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
                conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner)]));
                conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * input1_shared[(((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner) + 256)]));
                conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * input1_shared[((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner)]));
                conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * input1_shared[(((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner) + 256)]));
                conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * input1_shared[(((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner) + 8)]));
                conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * input1_shared[(((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner) + 264)]));
                conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * input1_shared[(((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner) + 8)]));
                conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((rc_outer_inner * 112) + (rc_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * input1_shared[(((((((int)threadIdx.x) / 14) * 16) + (rc_outer_inner * 4)) + rc_inner) + 264)]));
              }
            }
          }
          for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
            for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
              T_add[(((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (ax1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 14) * 2)) + ax3_inner)] = (conv2d_nchw[((ax1_inner * 2) + ax3_inner)] + input2[(((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (ax1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 14) * 2)) + ax3_inner)]);
              T_add[((((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (ax1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 14) * 2)) + ax3_inner) + 6272)] = (conv2d_nchw[(((ax1_inner * 2) + ax3_inner) + 4)] + input2[((((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (ax1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 14) * 2)) + ax3_inner) + 6272)]);
            }
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_575<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

