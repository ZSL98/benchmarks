#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_561
// Description:	Matched_Pattern
// Input:
//	- name: Relu_284_0	type: float	shape: Shape{8, 2048, 7, 7}
//	- name: Constant_514_0	type: float	shape: Shape{512, 2048, 1, 1}
//	- name: Broadcast_515_0	type: float	shape: Shape{8, 512, 7, 7}
// Output:
//	- name: Relu_288_0	type: float	shape: Shape{8, 512, 7, 7}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_561(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[224];
    __shared__ float input1_shared[1024];
    {
        float* compute = output0;
        {
          float conv2d_nchw[14];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
            __syncthreads();
            pad_temp_shared[((int)threadIdx.x)] = input0[((((((((int)blockIdx.x) / 56) * 200704) + (rc_outer_outer * 784)) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7))];
            pad_temp_shared[(((int)threadIdx.x) + 64)] = input0[(((((((((int)blockIdx.x) / 56) * 200704) + (((((int)threadIdx.x) + 64) / 112) * 100352)) + (rc_outer_outer * 784)) + ((((((int)threadIdx.x) + 64) % 112) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 1) % 7))];
            pad_temp_shared[(((int)threadIdx.x) + 128)] = input0[(((((((((int)blockIdx.x) / 56) * 200704) + (((((int)threadIdx.x) + 128) / 112) * 100352)) + (rc_outer_outer * 784)) + (((((int)threadIdx.x) + 16) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 2) % 7))];
            if (((int)threadIdx.x) < 32) {
              pad_temp_shared[(((int)threadIdx.x) + 192)] = input0[(((((((((int)blockIdx.x) / 56) * 200704) + (((((int)threadIdx.x) + 192) / 112) * 100352)) + (rc_outer_outer * 784)) + (((((int)threadIdx.x) + 80) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 3) % 7))];
            }
            *(float2*)(input1_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input1 + ((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 128)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 16384));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 32768));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 384)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 49152));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 65536));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 640)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 81920));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 98304));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 896)) = *(float2*)(input1 + (((((((((int)blockIdx.x) % 56) / 7) * 131072) + ((((int)threadIdx.x) >> 3) * 2048)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 114688));
            __syncthreads();
            for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
              for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[((nn_outer_inner * 112) + xx_outer_inner)] * input1_shared[(((int)threadIdx.x) * 16)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 7)] * input1_shared[((((int)threadIdx.x) * 16) + 1)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 14)] * input1_shared[((((int)threadIdx.x) * 16) + 2)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 21)] * input1_shared[((((int)threadIdx.x) * 16) + 3)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 28)] * input1_shared[((((int)threadIdx.x) * 16) + 4)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 35)] * input1_shared[((((int)threadIdx.x) * 16) + 5)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 42)] * input1_shared[((((int)threadIdx.x) * 16) + 6)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 49)] * input1_shared[((((int)threadIdx.x) * 16) + 7)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 56)] * input1_shared[((((int)threadIdx.x) * 16) + 8)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 63)] * input1_shared[((((int)threadIdx.x) * 16) + 9)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 70)] * input1_shared[((((int)threadIdx.x) * 16) + 10)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 77)] * input1_shared[((((int)threadIdx.x) * 16) + 11)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 84)] * input1_shared[((((int)threadIdx.x) * 16) + 12)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 91)] * input1_shared[((((int)threadIdx.x) * 16) + 13)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 98)] * input1_shared[((((int)threadIdx.x) * 16) + 14)]));
                conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((nn_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[(((nn_outer_inner * 112) + xx_outer_inner) + 105)] * input1_shared[((((int)threadIdx.x) * 16) + 15)]));
              }
            }
          }
          for (int i0_inner = 0; i0_inner < 2; ++i0_inner) {
            for (int i3_inner = 0; i3_inner < 7; ++i3_inner) {
              compute[(((((((((int)blockIdx.x) / 56) * 50176) + (i0_inner * 25088)) + (((((int)blockIdx.x) % 56) / 7) * 3136)) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + i3_inner)] = max((conv2d_nchw[((i0_inner * 7) + i3_inner)] + input2[(((((((((int)blockIdx.x) / 56) * 50176) + (i0_inner * 25088)) + (((((int)blockIdx.x) % 56) / 7) * 3136)) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + i3_inner)]), 0.000000e+00f);
            }
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_561_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_561<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

