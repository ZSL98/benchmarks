#include "shared.h"
// Node name:	Constant_433
// Description:	Constant
// Input:
// Output:
//	- name: Constant_433_0	type: float	shape: Shape{512, 128, 1, 1}
void Constant_float_cuda_Constant_433(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_433_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_433_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[262144];
    bin_file.read(tmp_mem, 262144);
    hipMemcpyAsync(output0, tmp_mem, 262144, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

