#include "shared.h"
// Node name:	Constant_502
// Description:	Constant
// Input:
// Output:
//	- name: Constant_502_0	type: float	shape: Shape{512, 1024, 1, 1}
void Constant_float_cuda_Constant_502(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_502_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_502_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2097152];
    bin_file.read(tmp_mem, 2097152);
    hipMemcpyAsync(output0, tmp_mem, 2097152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

