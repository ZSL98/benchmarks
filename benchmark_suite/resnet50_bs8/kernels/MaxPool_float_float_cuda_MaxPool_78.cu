#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	MaxPool_78
// Description:	MaxPool
// Input:
//	- name: Relu_77_0	type: float	shape: Shape{8, 64, 112, 112}
// Output:
//	- name: MaxPool_78_0	type: float	shape: Shape{8, 64, 56, 56}
extern "C" __global__  void MaxPool_float_float_cuda_MaxPool_78(float* input0, float* output0)
{
    {
        float* data = output0;
        float* tensor = input0;
        {
          tensor[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))] = -3.402823e+38f;
          for (int rv0 = 0; rv0 < 3; ++rv0) {
            for (int rv1 = 0; rv1 < 3; ++rv1) {
              tensor[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))], data[((((((((int)blockIdx.x) / 55) * 12544) + ((((int)blockIdx.x) % 55) * 224)) + (rv0 * 112)) + (((int)threadIdx.x) * 2)) + rv1)]);
            }
          }
        }

    }

}
extern void MaxPool_float_float_cuda_MaxPool_78_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* output0) {
    MaxPool_float_float_cuda_MaxPool_78<<<grids, blocks, mem, stream>>>(input0, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

