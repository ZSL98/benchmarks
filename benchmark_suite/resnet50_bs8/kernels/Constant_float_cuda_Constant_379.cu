#include "shared.h"
// Node name:	Constant_379
// Description:	Constant
// Input:
// Output:
//	- name: Constant_379_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_379(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_379_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_379_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

