#include "shared.h"
// Node name:	Constant_475
// Description:	Constant
// Input:
// Output:
//	- name: Constant_475_0	type: float	shape: Shape{256, 1024, 1, 1}
void Constant_float_cuda_Constant_475(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_475_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_475_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1048576];
    bin_file.read(tmp_mem, 1048576);
    hipMemcpyAsync(output0, tmp_mem, 1048576, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

