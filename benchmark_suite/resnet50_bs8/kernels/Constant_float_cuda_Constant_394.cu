#include "shared.h"
// Node name:	Constant_394
// Description:	Constant
// Input:
// Output:
//	- name: Constant_394_0	type: float	shape: Shape{256, 64, 1, 1}
void Constant_float_cuda_Constant_394(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_394_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_394_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

