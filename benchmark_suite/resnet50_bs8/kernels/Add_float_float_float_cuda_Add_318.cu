#include "hip/hip_runtime.h"
#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
// Node name:	Add_318
// Description:	Add
// Input:
//	- name: Dot_316_0	type: float	shape: Shape{8, 1000}
//	- name: Broadcast_317_0	type: float	shape: Shape{8, 1000}
// Output:
//	- name: Add_318_0	type: float	shape: Shape{8, 1000}
extern "C" __launch_bounds__(64) __global__ void Add_float_float_float_cuda_Add_318(float* input0, float* input1, float* output0)
{
    output0[blockIdx.x * 64 * 2 + threadIdx.x] = add(input0[blockIdx.x * 64 * 2 + threadIdx.x], input1[blockIdx.x * 64 * 2 + threadIdx.x]);
    output0[blockIdx.x * 64 * 2 + threadIdx.x + 128] = add(input0[blockIdx.x * 64 * 2 + threadIdx.x + 128], input1[blockIdx.x * 64 * 2 + threadIdx.x + 128]);

}
extern void Add_float_float_float_cuda_Add_318_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Add_float_float_float_cuda_Add_318<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

