#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_569
// Description:	Matched_Pattern
// Input:
//	- name: Pad_122_0	type: float	shape: Shape{8, 256, 56, 56}
//	- name: Constant_409_0	type: float	shape: Shape{512, 256, 1, 1}
//	- name: Broadcast_410_0	type: float	shape: Shape{8, 512, 28, 28}
// Output:
//	- name: BatchNormInference_128_0	type: float	shape: Shape{8, 512, 28, 28}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_569(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[1760];
    __shared__ float input1_shared[1024];
    {
        float* T_add = output0;
        {
          float conv2d_nchw[16];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[14] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          conv2d_nchw[15] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
            __syncthreads();
            pad_temp_shared[((int)threadIdx.x)] = input0[((((((((int)blockIdx.x) / 224) * 1605632) + (rc_outer_outer * 50176)) + ((((int)threadIdx.x) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) % 55))];
            pad_temp_shared[(((int)threadIdx.x) + 224)] = input0[((((((((int)blockIdx.x) / 224) * 1605632) + (rc_outer_outer * 50176)) + (((((int)threadIdx.x) + 224) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 4) % 55))];
            pad_temp_shared[(((int)threadIdx.x) + 448)] = input0[((((((((int)blockIdx.x) / 224) * 1605632) + (rc_outer_outer * 50176)) + (((((int)threadIdx.x) + 448) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 8) % 55))];
            pad_temp_shared[(((int)threadIdx.x) + 672)] = input0[(((((((((int)blockIdx.x) / 224) * 1605632) + (((((int)threadIdx.x) + 672) / 880) * 802816)) + (rc_outer_outer * 50176)) + ((((((int)threadIdx.x) + 672) % 880) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 12) % 55))];
            pad_temp_shared[(((int)threadIdx.x) + 896)] = input0[(((((((((int)blockIdx.x) / 224) * 1605632) + (((((int)threadIdx.x) + 896) / 880) * 802816)) + (rc_outer_outer * 50176)) + (((((int)threadIdx.x) + 16) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 16) % 55))];
            pad_temp_shared[(((int)threadIdx.x) + 1120)] = input0[(((((((((int)blockIdx.x) / 224) * 1605632) + (((((int)threadIdx.x) + 1120) / 880) * 802816)) + (rc_outer_outer * 50176)) + (((((int)threadIdx.x) + 240) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 20) % 55))];
            pad_temp_shared[(((int)threadIdx.x) + 1344)] = input0[(((((((((int)blockIdx.x) / 224) * 1605632) + (((((int)threadIdx.x) + 1344) / 880) * 802816)) + (rc_outer_outer * 50176)) + (((((int)threadIdx.x) + 464) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 24) % 55))];
            if (((int)threadIdx.x) < 192) {
              pad_temp_shared[(((int)threadIdx.x) + 1568)] = input0[(((((((((int)blockIdx.x) / 224) * 1605632) + (((((int)threadIdx.x) + 1568) / 880) * 802816)) + (rc_outer_outer * 50176)) + (((((int)threadIdx.x) + 688) / 55) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 28) % 55))];
            }
            input1_shared[((int)threadIdx.x)] = input1[((((((((int)blockIdx.x) % 224) / 28) * 16384) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
            input1_shared[(((int)threadIdx.x) + 224)] = input1[(((((((((int)blockIdx.x) % 224) / 28) * 16384) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 3584)];
            input1_shared[(((int)threadIdx.x) + 448)] = input1[(((((((((int)blockIdx.x) % 224) / 28) * 16384) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 7168)];
            input1_shared[(((int)threadIdx.x) + 672)] = input1[(((((((((int)blockIdx.x) % 224) / 28) * 16384) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 10752)];
            if (((int)threadIdx.x) < 128) {
              input1_shared[(((int)threadIdx.x) + 896)] = input1[(((((((((int)blockIdx.x) % 224) / 28) * 16384) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 14336)];
            }
            __syncthreads();
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 14) * 2)] * input1_shared[((((int)threadIdx.x) / 14) * 16)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 28)] * input1_shared[((((int)threadIdx.x) / 14) * 16)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((int)threadIdx.x) % 14) * 2)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 256)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 28)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 256)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((int)threadIdx.x) % 14) * 2)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 512)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 28)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 512)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((int)threadIdx.x) % 14) * 2)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 768)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 28)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 768)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 880)] * input1_shared[((((int)threadIdx.x) / 14) * 16)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 908)] * input1_shared[((((int)threadIdx.x) / 14) * 16)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 880)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 256)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 908)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 256)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 880)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 512)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 908)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 512)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 880)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 768)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 908)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 768)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 55)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 1)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 83)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 1)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 55)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 257)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 83)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 257)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 55)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 513)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 83)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 513)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 55)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 769)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 83)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 769)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 935)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 1)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 963)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 1)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 935)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 257)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 963)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 257)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 935)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 513)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 963)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 513)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 935)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 769)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 963)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 769)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 110)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 2)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 138)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 2)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 110)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 258)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 138)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 258)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 110)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 514)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 138)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 514)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 110)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 770)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 138)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 770)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 990)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 2)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1018)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 2)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 990)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 258)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1018)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 258)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 990)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 514)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1018)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 514)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 990)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 770)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1018)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 770)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 165)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 3)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 193)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 3)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 165)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 259)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 193)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 259)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 165)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 515)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 193)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 515)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 165)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 771)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 193)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 771)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1045)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 3)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1073)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 3)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1045)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 259)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1073)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 259)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1045)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 515)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1073)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 515)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1045)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 771)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1073)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 771)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 220)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 4)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 248)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 4)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 220)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 260)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 248)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 260)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 220)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 516)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 248)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 516)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 220)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 772)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 248)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 772)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1100)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 4)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1128)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 4)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1100)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 260)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1128)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 260)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1100)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 516)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1128)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 516)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1100)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 772)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1128)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 772)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 275)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 5)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 303)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 5)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 275)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 261)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 303)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 261)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 275)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 517)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 303)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 517)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 275)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 773)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 303)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 773)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1155)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 5)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1183)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 5)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1155)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 261)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1183)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 261)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1155)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 517)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1183)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 517)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1155)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 773)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1183)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 773)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 330)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 6)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 358)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 6)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 330)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 262)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 358)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 262)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 330)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 518)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 358)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 518)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 330)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 774)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 358)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 774)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1210)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 6)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1238)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 6)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1210)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 262)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1238)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 262)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1210)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 518)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1238)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 518)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1210)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 774)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1238)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 774)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 385)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 7)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 413)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 7)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 385)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 263)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 413)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 263)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 385)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 519)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 413)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 519)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 385)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 775)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 413)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 775)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1265)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 7)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1293)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 7)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1265)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 263)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1293)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 263)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1265)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 519)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1293)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 519)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1265)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 775)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1293)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 775)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 440)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 8)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 468)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 8)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 440)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 264)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 468)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 264)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 440)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 520)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 468)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 520)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 440)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 776)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 468)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 776)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1320)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 8)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1348)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 8)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1320)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 264)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1348)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 264)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1320)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 520)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1348)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 520)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1320)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 776)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1348)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 776)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 495)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 9)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 523)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 9)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 495)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 265)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 523)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 265)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 495)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 521)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 523)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 521)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 495)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 777)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 523)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 777)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1375)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 9)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1403)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 9)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1375)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 265)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1403)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 265)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1375)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 521)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1403)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 521)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1375)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 777)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1403)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 777)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 550)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 10)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 578)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 10)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 550)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 266)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 578)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 266)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 550)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 522)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 578)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 522)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 550)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 778)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 578)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 778)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1430)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 10)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1458)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 10)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1430)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 266)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1458)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 266)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1430)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 522)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1458)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 522)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1430)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 778)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1458)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 778)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 605)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 11)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 633)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 11)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 605)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 267)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 633)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 267)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 605)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 523)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 633)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 523)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 605)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 779)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 633)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 779)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1485)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 11)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1513)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 11)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1485)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 267)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1513)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 267)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1485)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 523)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1513)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 523)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1485)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 779)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1513)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 779)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 660)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 12)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 688)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 12)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 660)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 268)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 688)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 268)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 660)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 524)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 688)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 524)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 660)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 780)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 688)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 780)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1540)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 12)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1568)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 12)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1540)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 268)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1568)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 268)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1540)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 524)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1568)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 524)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1540)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 780)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1568)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 780)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 715)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 13)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 743)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 13)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 715)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 269)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 743)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 269)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 715)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 525)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 743)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 525)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 715)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 781)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 743)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 781)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1595)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 13)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1623)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 13)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1595)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 269)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1623)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 269)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1595)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 525)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1623)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 525)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1595)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 781)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1623)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 781)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 770)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 14)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 798)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 14)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 770)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 270)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 798)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 270)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 770)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 526)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 798)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 526)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 770)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 782)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 798)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 782)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1650)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 14)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1678)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 14)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1650)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 270)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1678)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 270)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1650)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 526)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1678)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 526)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1650)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 782)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1678)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 782)]));
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 825)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 15)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 853)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 15)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 825)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 271)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 853)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 271)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 825)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 527)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 853)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 527)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 825)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 783)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 853)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 783)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1705)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 15)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1733)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 15)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1705)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 271)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1733)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 271)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1705)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 527)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1733)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 527)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1705)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 783)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((int)threadIdx.x) % 14) * 2) + 1733)] * input1_shared[(((((int)threadIdx.x) / 14) * 16) + 783)]));
          }
          for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
            T_add[(((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14))] = (conv2d_nchw[ax0_inner] + input2[(((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14))]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 14)] = (conv2d_nchw[(ax0_inner + 2)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 14)]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 12544)] = (conv2d_nchw[(ax0_inner + 4)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 12544)]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 12558)] = (conv2d_nchw[(ax0_inner + 6)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 12558)]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 25088)] = (conv2d_nchw[(ax0_inner + 8)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 25088)]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 25102)] = (conv2d_nchw[(ax0_inner + 10)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 25102)]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 37632)] = (conv2d_nchw[(ax0_inner + 12)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 37632)]);
            T_add[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 37646)] = (conv2d_nchw[(ax0_inner + 14)] + input2[((((((((((int)blockIdx.x) / 224) * 802816) + (ax0_inner * 401408)) + (((((int)blockIdx.x) % 224) / 28) * 50176)) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 37646)]);
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_569_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_569<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

