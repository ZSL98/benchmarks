#include "shared.h"
// Node name:	Constant_312
// Description:	Constant
// Input:
// Output:
//	- name: Constant_312_0	type: float	shape: Shape{8, 2048}
void Constant_float_cuda_Constant_312(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_312_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_312_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

