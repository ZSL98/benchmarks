#include "shared.h"
// Node name:	Constant_54
// Description:	Constant
// Input:
// Output:
//	- name: Constant_54_0	type: float	shape: Shape{2048}
void Constant_float_cuda_Constant_54(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_54_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_54_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8192];
    bin_file.read(tmp_mem, 8192);
    hipMemcpyAsync(output0, tmp_mem, 8192, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

