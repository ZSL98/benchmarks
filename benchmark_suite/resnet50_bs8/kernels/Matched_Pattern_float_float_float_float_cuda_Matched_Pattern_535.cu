#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Matched_Pattern_535
// Description:	Matched_Pattern
// Input:
//	- name: Relu_94_0	type: float	shape: Shape{8, 256, 56, 56}
//	- name: Constant_388_0	type: float	shape: Shape{64, 256, 1, 1}
//	- name: Broadcast_389_0	type: float	shape: Shape{8, 64, 56, 56}
// Output:
//	- name: Relu_98_0	type: float	shape: Shape{8, 64, 56, 56}
extern "C" __global__  void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_535(float* input0, float* input1, float* input2, float* output0)
{
    __shared__ float pad_temp_shared[2048];
    __shared__ float input1_shared[2048];
    {
        float* compute = output0;
        {
          float conv2d_nchw[16];
          
          
          conv2d_nchw[0] = 0.000000e+00f;
          conv2d_nchw[8] = 0.000000e+00f;
          conv2d_nchw[1] = 0.000000e+00f;
          conv2d_nchw[9] = 0.000000e+00f;
          conv2d_nchw[2] = 0.000000e+00f;
          conv2d_nchw[10] = 0.000000e+00f;
          conv2d_nchw[3] = 0.000000e+00f;
          conv2d_nchw[11] = 0.000000e+00f;
          conv2d_nchw[4] = 0.000000e+00f;
          conv2d_nchw[12] = 0.000000e+00f;
          conv2d_nchw[5] = 0.000000e+00f;
          conv2d_nchw[13] = 0.000000e+00f;
          conv2d_nchw[6] = 0.000000e+00f;
          conv2d_nchw[14] = 0.000000e+00f;
          conv2d_nchw[7] = 0.000000e+00f;
          conv2d_nchw[15] = 0.000000e+00f;
          for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
            __syncthreads();
            pad_temp_shared[((int)threadIdx.x)] = input0[((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
            pad_temp_shared[(((int)threadIdx.x) + 256)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 25088)];
            pad_temp_shared[(((int)threadIdx.x) + 512)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 50176)];
            pad_temp_shared[(((int)threadIdx.x) + 768)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 75264)];
            pad_temp_shared[(((int)threadIdx.x) + 1024)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 802816)];
            pad_temp_shared[(((int)threadIdx.x) + 1280)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 827904)];
            pad_temp_shared[(((int)threadIdx.x) + 1536)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 852992)];
            pad_temp_shared[(((int)threadIdx.x) + 1792)] = input0[(((((((((((int)blockIdx.x) / 98) * 1605632) + (rc_outer_outer * 100352)) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 878080)];
            *(float2*)(input1_shared + (((int)threadIdx.x) * 2)) = *(float2*)(input1 + ((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 4096));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8192));
            *(float2*)(input1_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(input1 + (((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 12288));
            __syncthreads();
            for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
              conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * input1_shared[(((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner)]));
              conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1024)]));
              conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * input1_shared[(((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner)]));
              conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1024)]));
              conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 32)]));
              conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1056)]));
              conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 32)]));
              conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1056)]));
              conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1024)] * input1_shared[(((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner)]));
              conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1024)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1024)]));
              conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1032)] * input1_shared[(((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner)]));
              conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1032)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1024)]));
              conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1024)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 32)]));
              conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1024)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1056)]));
              conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1032)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 32)]));
              conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[((((rc_outer_inner * 32) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 1032)] * input1_shared[((((((int)threadIdx.x) >> 4) * 64) + rc_outer_inner) + 1056)]));
            }
          }
          for (int i0_inner = 0; i0_inner < 2; ++i0_inner) {
            for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
              for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
                compute[((((((((((((int)blockIdx.x) / 98) * 401408) + (i0_inner * 200704)) + ((((int)threadIdx.x) >> 4) * 6272)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = max((conv2d_nchw[(((i0_inner * 4) + (i1_inner * 2)) + i2_inner)] + input2[((((((((((((int)blockIdx.x) / 98) * 401408) + (i0_inner * 200704)) + ((((int)threadIdx.x) >> 4) * 6272)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))]), 0.000000e+00f);
                compute[(((((((((((((int)blockIdx.x) / 98) * 401408) + (i0_inner * 200704)) + ((((int)threadIdx.x) >> 4) * 6272)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 100352)] = max((conv2d_nchw[((((i0_inner * 4) + (i1_inner * 2)) + i2_inner) + 8)] + input2[(((((((((((((int)blockIdx.x) / 98) * 401408) + (i0_inner * 200704)) + ((((int)threadIdx.x) >> 4) * 6272)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 98) / 7) * 224)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 100352)]), 0.000000e+00f);
              }
            }
          }
        }

    }

}
extern void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_535_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_535<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

