#include "shared.h"
// Node name:	Constant_424
// Description:	Constant
// Input:
// Output:
//	- name: Constant_424_0	type: float	shape: Shape{512, 128, 1, 1}
void Constant_float_cuda_Constant_424(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_424_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_424_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[262144];
    bin_file.read(tmp_mem, 262144);
    hipMemcpyAsync(output0, tmp_mem, 262144, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

