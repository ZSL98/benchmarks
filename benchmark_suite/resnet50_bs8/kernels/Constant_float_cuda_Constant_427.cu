#include "shared.h"
// Node name:	Constant_427
// Description:	Constant
// Input:
// Output:
//	- name: Constant_427_0	type: float	shape: Shape{128, 512, 1, 1}
void Constant_float_cuda_Constant_427(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_427_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_427_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[262144];
    bin_file.read(tmp_mem, 262144);
    hipMemcpyAsync(output0, tmp_mem, 262144, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

