#include "shared.h"
// Node name:	Constant_376
// Description:	Constant
// Input:
// Output:
//	- name: Constant_376_0	type: float	shape: Shape{256, 64, 1, 1}
void Constant_float_cuda_Constant_376(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_376_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_376_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

