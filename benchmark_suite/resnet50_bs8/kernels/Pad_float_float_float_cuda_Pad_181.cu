#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Pad_181
// Description:	Pad
// Input:
//	- name: Relu_179_0	type: float	shape: Shape{8, 512, 28, 28}
//	- name: Constant_180_0	type: float	shape: Shape{}
// Output:
//	- name: Pad_181_0	type: float	shape: Shape{8, 512, 28, 28}
extern "C" __launch_bounds__(64) __global__ void Pad_float_float_float_cuda_Pad_181(float* input0, float* input1, float* output0)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float* in = input0;
    float* pad = input1;
    float* out = output0;
    if (tid < 3211264)
    {
        size_t input_shape0 = 8;
        size_t input_shape1 = 512;
        size_t input_shape2 = 28;
        size_t input_shape3 = 28;
        uint32_t input_strides0 = 401408;
        uint32_t input_strides1 = 784;
        uint32_t input_strides2 = 28;
        uint32_t input_strides3 = 1;
        uint32_t output_strides0 = 401408;
        uint32_t output_strides1 = 784;
        uint32_t output_strides2 = 28;
        uint32_t output_strides3 = 1;
        uint32_t padding_below0 = 0;
        uint32_t padding_below1 = 0;
        uint32_t padding_below2 = 0;
        uint32_t padding_below3 = 0;
        uint32_t padding_interior0 = 0;
        uint32_t padding_interior1 = 0;
        uint32_t padding_interior2 = 0;
        uint32_t padding_interior3 = 0;
        bool in_bounds = true;
        uint32_t output_pixel = tid;
        uint32_t input_pixel = 0;
        int32_t input, input_dil;
        input_dil = output_pixel / output_strides0 - padding_below0;
        input = input_dil / (padding_interior0 + 1);
        input_dil %= (padding_interior0 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
        input_pixel += input * input_strides0;
        output_pixel %= output_strides0;
        input_dil = output_pixel / output_strides1 - padding_below1;
        input = input_dil / (padding_interior1 + 1);
        input_dil %= (padding_interior1 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
        input_pixel += input * input_strides1;
        output_pixel %= output_strides1;
        input_dil = output_pixel / output_strides2 - padding_below2;
        input = input_dil / (padding_interior2 + 1);
        input_dil %= (padding_interior2 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
        input_pixel += input * input_strides2;
        output_pixel %= output_strides2;
        input_dil = output_pixel / output_strides3 - padding_below3;
        input = input_dil / (padding_interior3 + 1);
        input_dil %= (padding_interior3 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
        input_pixel += input * input_strides3;
        out[tid] = (in_bounds) ? in[input_pixel] : *pad;
    }

}
extern void Pad_float_float_float_cuda_Pad_181_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Pad_float_float_float_cuda_Pad_181<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

