#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Pad_275
// Description:	Pad
// Input:
//	- name: Relu_272_0	type: float	shape: Shape{8, 512, 14, 14}
//	- name: Constant_274_0	type: float	shape: Shape{}
// Output:
//	- name: Pad_275_0	type: float	shape: Shape{8, 512, 16, 16}
extern "C" __launch_bounds__(64) __global__ void Pad_float_float_float_cuda_Pad_275(float* input0, float* input1, float* output0)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float* in = input0;
    float* pad = input1;
    float* out = output0;
    if (tid < 1048576)
    {
        size_t input_shape0 = 8;
        size_t input_shape1 = 512;
        size_t input_shape2 = 14;
        size_t input_shape3 = 14;
        uint32_t input_strides0 = 100352;
        uint32_t input_strides1 = 196;
        uint32_t input_strides2 = 14;
        uint32_t input_strides3 = 1;
        uint32_t output_strides0 = 131072;
        uint32_t output_strides1 = 256;
        uint32_t output_strides2 = 16;
        uint32_t output_strides3 = 1;
        uint32_t padding_below0 = 0;
        uint32_t padding_below1 = 0;
        uint32_t padding_below2 = 1;
        uint32_t padding_below3 = 1;
        uint32_t padding_interior0 = 0;
        uint32_t padding_interior1 = 0;
        uint32_t padding_interior2 = 0;
        uint32_t padding_interior3 = 0;
        bool in_bounds = true;
        uint32_t output_pixel = tid;
        uint32_t input_pixel = 0;
        int32_t input, input_dil;
        input_dil = output_pixel / output_strides0 - padding_below0;
        input = input_dil / (padding_interior0 + 1);
        input_dil %= (padding_interior0 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
        input_pixel += input * input_strides0;
        output_pixel %= output_strides0;
        input_dil = output_pixel / output_strides1 - padding_below1;
        input = input_dil / (padding_interior1 + 1);
        input_dil %= (padding_interior1 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
        input_pixel += input * input_strides1;
        output_pixel %= output_strides1;
        input_dil = output_pixel / output_strides2 - padding_below2;
        input = input_dil / (padding_interior2 + 1);
        input_dil %= (padding_interior2 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
        input_pixel += input * input_strides2;
        output_pixel %= output_strides2;
        input_dil = output_pixel / output_strides3 - padding_below3;
        input = input_dil / (padding_interior3 + 1);
        input_dil %= (padding_interior3 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
        input_pixel += input * input_strides3;
        out[tid] = (in_bounds) ? in[input_pixel] : *pad;
    }

}
extern void Pad_float_float_float_cuda_Pad_275_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Pad_float_float_float_cuda_Pad_275<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

