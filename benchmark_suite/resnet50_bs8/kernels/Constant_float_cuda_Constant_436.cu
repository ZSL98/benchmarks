#include "shared.h"
// Node name:	Constant_436
// Description:	Constant
// Input:
// Output:
//	- name: Constant_436_0	type: float	shape: Shape{128, 512, 1, 1}
void Constant_float_cuda_Constant_436(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_436_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_436_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[262144];
    bin_file.read(tmp_mem, 262144);
    hipMemcpyAsync(output0, tmp_mem, 262144, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

