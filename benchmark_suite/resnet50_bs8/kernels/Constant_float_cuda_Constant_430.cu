#include "shared.h"
// Node name:	Constant_430
// Description:	Constant
// Input:
// Output:
//	- name: Constant_430_0	type: float	shape: Shape{128, 128, 3, 3}
void Constant_float_cuda_Constant_430(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_430_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_430_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[589824];
    bin_file.read(tmp_mem, 589824);
    hipMemcpyAsync(output0, tmp_mem, 589824, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

