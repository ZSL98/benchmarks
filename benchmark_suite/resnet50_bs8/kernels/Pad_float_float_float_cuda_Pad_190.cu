#include "hip/hip_runtime.h"
#include "shared.h"
// Node name:	Pad_190
// Description:	Pad
// Input:
//	- name: Relu_188_0	type: float	shape: Shape{8, 256, 28, 28}
//	- name: Constant_189_0	type: float	shape: Shape{}
// Output:
//	- name: Pad_190_0	type: float	shape: Shape{8, 256, 30, 30}
extern "C" __launch_bounds__(64) __global__ void Pad_float_float_float_cuda_Pad_190(float* input0, float* input1, float* output0)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float* in = input0;
    float* pad = input1;
    float* out = output0;
    if (tid < 1843200)
    {
        size_t input_shape0 = 8;
        size_t input_shape1 = 256;
        size_t input_shape2 = 28;
        size_t input_shape3 = 28;
        uint32_t input_strides0 = 200704;
        uint32_t input_strides1 = 784;
        uint32_t input_strides2 = 28;
        uint32_t input_strides3 = 1;
        uint32_t output_strides0 = 230400;
        uint32_t output_strides1 = 900;
        uint32_t output_strides2 = 30;
        uint32_t output_strides3 = 1;
        uint32_t padding_below0 = 0;
        uint32_t padding_below1 = 0;
        uint32_t padding_below2 = 1;
        uint32_t padding_below3 = 1;
        uint32_t padding_interior0 = 0;
        uint32_t padding_interior1 = 0;
        uint32_t padding_interior2 = 0;
        uint32_t padding_interior3 = 0;
        bool in_bounds = true;
        uint32_t output_pixel = tid;
        uint32_t input_pixel = 0;
        int32_t input, input_dil;
        input_dil = output_pixel / output_strides0 - padding_below0;
        input = input_dil / (padding_interior0 + 1);
        input_dil %= (padding_interior0 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
        input_pixel += input * input_strides0;
        output_pixel %= output_strides0;
        input_dil = output_pixel / output_strides1 - padding_below1;
        input = input_dil / (padding_interior1 + 1);
        input_dil %= (padding_interior1 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
        input_pixel += input * input_strides1;
        output_pixel %= output_strides1;
        input_dil = output_pixel / output_strides2 - padding_below2;
        input = input_dil / (padding_interior2 + 1);
        input_dil %= (padding_interior2 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
        input_pixel += input * input_strides2;
        output_pixel %= output_strides2;
        input_dil = output_pixel / output_strides3 - padding_below3;
        input = input_dil / (padding_interior3 + 1);
        input_dil %= (padding_interior3 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
        input_pixel += input * input_strides3;
        out[tid] = (in_bounds) ? in[input_pixel] : *pad;
    }

}
extern void Pad_float_float_float_cuda_Pad_190_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Pad_float_float_float_cuda_Pad_190<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

