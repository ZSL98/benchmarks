#include "shared.h"
// Node name:	Constant_463
// Description:	Constant
// Input:
// Output:
//	- name: Constant_463_0	type: float	shape: Shape{1024, 256, 1, 1}
void Constant_float_cuda_Constant_463(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_463_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_463_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1048576];
    bin_file.read(tmp_mem, 1048576);
    hipMemcpyAsync(output0, tmp_mem, 1048576, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

