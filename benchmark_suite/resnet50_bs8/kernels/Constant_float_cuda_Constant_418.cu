#include "shared.h"
// Node name:	Constant_418
// Description:	Constant
// Input:
// Output:
//	- name: Constant_418_0	type: float	shape: Shape{128, 512, 1, 1}
void Constant_float_cuda_Constant_418(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_418_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_418_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[262144];
    bin_file.read(tmp_mem, 262144);
    hipMemcpyAsync(output0, tmp_mem, 262144, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

